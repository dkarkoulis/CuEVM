// cuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation Programme)
// Author: Stefan-Dan Ciocirlan
// Data: 2024-06-20
// SPDX-License-Identifier: MIT
#include "include/block.cuh"
#include "include/error_codes.h"
#include "include/utils.cuh"

namespace cuEVM {
  namespace block {
    __host__ __device__ EVMBlockInfo::EVMBlockInfo(
        ArithEnv arith,
        block_data_t *content
    ) : _arith(arith),
        content(content)
    {
    }
    __host__ EVMBlockInfo::EVMBlockInfo(
        ArithEnv arith,
        const cJSON *test
    ) : _arith(arith)
    {
      cJSON *block_json = NULL;
      cJSON *element_json = NULL;
      cJSON *previous_blocks_json = NULL;
      size_t idx = 0;
      content = NULL;
    #ifndef ONLY_CPU
      CUDA_CHECK(hipMallocManaged(
          (void **)&(content),
          sizeof(block_data_t)));
    #else
      content = new block_data_t;
    #endif

      block_json = cJSON_GetObjectItemCaseSensitive(test, "env");

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentCoinbase");
      content->coin_base.from_hex(element_json->valuestring);

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentTimestamp");
      content->time_stamp.from_hex(element_json->valuestring);

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentNumber");
      content->number.from_hex(element_json->valuestring);
  
      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentDifficulty");
      content->difficulty.from_hex(element_json->valuestring);

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentRandom");
      if (element_json != NULL)
      {
        content->prevrandao.from_hex(element_json->valuestring);
      }

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentGasLimit");
      content->gas_limit.from_hex(element_json->valuestring);

      // element_json=cJSON_GetObjectItemCaseSensitive(block_json, "currentChainId");
      //_arith.cgbn_memory_from_hex_string(content->chain_id, element_json->valuestring);
      content->chain_id.from_size_t(1);

      element_json = cJSON_GetObjectItemCaseSensitive(block_json, "currentBaseFee");
      content->base_fee.from_hex(element_json->valuestring);

      previous_blocks_json = cJSON_GetObjectItemCaseSensitive(block_json, "previousHashes");
      if (previous_blocks_json != NULL and cJSON_IsArray(previous_blocks_json))
      {
        idx = 0;
        cJSON_ArrayForEach(element_json, previous_blocks_json)
        {
          element_json = cJSON_GetObjectItemCaseSensitive(element_json, "number");
          content->previous_blocks[idx].number.from_hex(element_json->valuestring);

          element_json = cJSON_GetObjectItemCaseSensitive(element_json, "hash");
          content->previous_blocks[idx].hash.from_hex(element_json->valuestring);
          idx++;
        }
      }
      else
      {
        idx = 0;
        // TODO: maybe fill with something else
        _arith.cgbn_memory_from_size_t(content->previous_blocks[0].number, 0);

        element_json = cJSON_GetObjectItemCaseSensitive(block_json, "previousHash");

        if (element_json != NULL){
          _arith.cgbn_memory_from_hex_string(content->previous_blocks[0].hash, element_json->valuestring);
        } else {
          _arith.cgbn_memory_from_size_t(content->previous_blocks[0].hash, 0);
        }

        idx++;
      }

      // fill the remaing parents with 0
      for (size_t jdx = idx; jdx < 256; jdx++)
      {
        _arith.cgbn_memory_from_size_t(content->previous_blocks[jdx].number, 0);
        _arith.cgbn_memory_from_size_t(content->previous_blocks[jdx].hash, 0);
      }
    }


    __host__ __device__ EVMBlockInfo::~EVMBlockInfo()
    {
      content = NULL;
    }


    __host__ void EVMBlockInfo::free_content()
    {
    #ifndef ONLY_CPU
      CUDA_CHECK(hipFree(content));
    #else
      delete content;
    #endif
      content = NULL;
    }


    __host__ __device__ void EVMBlockInfo::get_coin_base(
        bn_t &coin_base)
    {
      cgbn_load(_arith.env, coin_base, &(content->coin_base));
    }


    __host__ __device__ void EVMBlockInfo::get_time_stamp(
        bn_t &time_stamp)
    {
      cgbn_load(_arith.env, time_stamp, &(content->time_stamp));
    }

    __host__ __device__ void EVMBlockInfo::get_number(
      bn_t &number)
    {
      cgbn_load(_arith.env, number, &(content->number));
    }


    __host__ __device__ void EVMBlockInfo::get_difficulty(
      bn_t &difficulty)
    {
      cgbn_load(_arith.env, difficulty, &(content->difficulty));
    }

    __host__ __device__ void EVMBlockInfo::get_prevrandao(
      bn_t &val)
    {
      cgbn_load(_arith.env, val, &(content->prevrandao));
    }


    __host__ __device__ void EVMBlockInfo::get_gas_limit(
      bn_t &gas_limit)
    {
      cgbn_load(_arith.env, gas_limit, &(content->gas_limit));
    }


    __host__ __device__ void EVMBlockInfo::get_chain_id(
      bn_t &chain_id)
    {
      cgbn_load(_arith.env, chain_id, &(content->chain_id));
    }


    __host__ __device__ void EVMBlockInfo::get_base_fee(
      bn_t &base_fee)
    {
      cgbn_load(_arith.env, base_fee, &(content->base_fee));
    }

    __host__ __device__ void EVMBlockInfo::get_previous_hash(
        bn_t &previous_hash,
        bn_t &previous_number,
        uint32_t &error_code)
    {
      uint32_t idx = 0;
      bn_t number;
      // ge tthe current number
      get_number(number);
      // if the rquest number is greater than the current block number
      if (cgbn_compare(_arith.env, number, previous_number) < 1)
      {
        error_code = ERR_BLOCK_INVALID_NUMBER;
      }
      // get the distance from the current block number to the requested block number
      cgbn_sub(_arith.env, number, number, previous_number);
      idx = cgbn_get_ui32(_arith.env, number) - 1;
      // only the last 256 blocks are stored
      if (idx > 255)
      {
        error_code = ERR_BLOCK_INVALID_NUMBER;
      }
      if (error_code == ERR_NONE)
        cgbn_load(_arith.env, previous_hash, &(content->previous_blocks[idx].hash));
      else
        cgbn_set_ui32(_arith.env, previous_hash, 0);
    }

    __host__ __device__ void EVMBlockInfo::print()
    {
      uint32_t idx = 0;
      bn_t number;
      printf("BLOCK: \n");
      printf("COINBASE: ");
      _arith.print_cgbn_memory(content->coin_base);
      printf("TIMESTAMP: ");
      _arith.print_cgbn_memory(content->time_stamp);
      printf("NUMBER: ");
      _arith.print_cgbn_memory(content->number);
      printf("DIFICULTY: ");
      _arith.print_cgbn_memory(content->difficulty);
      printf("GASLIMIT: ");
      _arith.print_cgbn_memory(content->gas_limit);
      printf("CHAINID: ");
      _arith.print_cgbn_memory(content->chain_id);
      printf("BASE_FEE: ");
      _arith.print_cgbn_memory(content->base_fee);
      printf("PREVIOUS_BLOCKS: \n");
      for (idx = 0; idx < 256; idx++)
      {
        printf("NUMBER: ");
        _arith.print_cgbn_memory(content->previous_blocks[idx].number);
        printf("HASH: ");
        _arith.print_cgbn_memory(content->previous_blocks[idx].hash);
        printf("\n");
        cgbn_load(_arith.env, number, &(content->previous_blocks[idx].number));
        if (cgbn_compare_ui32(_arith.env, number, 0) == 0)
        {
          break;
        }
      }
    }

    __host__ cJSON * EVMBlockInfo::to_json()
    {
      uint32_t idx = 0;
      char *hex_string_ptr = new char[EVM_WORD_SIZE * 2 + 3];
      cJSON *block_json = NULL;
      cJSON *previous_blocks_json = NULL;
      cJSON *previous_block_json = NULL;

      block_json = cJSON_CreateObject();

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->coin_base, 5);
      cJSON_AddStringToObject(block_json, "currentCoinbase", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->time_stamp);
      cJSON_AddStringToObject(block_json, "currentTimestamp", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->number);
      cJSON_AddStringToObject(block_json, "currentNumber", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->difficulty);
      cJSON_AddStringToObject(block_json, "currentDifficulty", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->gas_limit);
      cJSON_AddStringToObject(block_json, "currentGasLimit", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->chain_id);
      cJSON_AddStringToObject(block_json, "currentChainId", hex_string_ptr);

      _arith.hex_string_from_cgbn_memory(hex_string_ptr, content->base_fee);
      cJSON_AddStringToObject(block_json, "currentBaseFee", hex_string_ptr);

      previous_blocks_json = cJSON_CreateArray();
      bn_t number;
      for (idx = 0; idx < 256; idx++)
      {
        previous_block_json = cJSON_CreateObject();

        _arith.hex_string_from_cgbn_memory(
          hex_string_ptr,
          content->previous_blocks[idx].number
        );
        cJSON_AddStringToObject(previous_block_json, "number", hex_string_ptr);

        _arith.hex_string_from_cgbn_memory(
          hex_string_ptr,
          content->previous_blocks[idx].hash
        );
        cJSON_AddStringToObject(previous_block_json, "hash", hex_string_ptr);

        cJSON_AddItemToArray(previous_blocks_json, previous_block_json);

        cgbn_load(_arith.env, number, &(content->previous_blocks[idx].number));
        if (cgbn_compare_ui32(_arith.env, number, 0) == 0)
        {
          break;
        }
      }

      cJSON_AddItemToObject(block_json, "previousHashes", previous_blocks_json);
      
      delete[] hex_string_ptr;
      hex_string_ptr = NULL;
      return block_json;
    }
  }
}
