#include <getopt.h>
#include <fstream>

#include "utils.cu"
#include "evm.cuh"

void run_interpreter(cJSON *read_root, cJSON *write_root) {
  // typedef evm_t<evm_params> evm_t;
  typedef typename evm_t::evm_instances_t evm_instances_t;
  typedef arith_env_t<evm_params> arith_t;

  evm_instances_t         cpu_instances;
  #ifndef ONLY_CPU
  evm_instances_t tmp_gpu_instances, *gpu_instances;
  cgbn_error_report_t     *report;
  CUDA_CHECK(hipDeviceReset());
  #endif

  arith_t arith(cgbn_report_monitor, 0);
  
  if(read_root == NULL) {
    printf("Error: could not read the json file\n");
    exit(EXIT_FAILURE);
  }
  
  const cJSON *test = NULL;
  cJSON_ArrayForEach(test, read_root) {
    // get instaces to run
    printf("Generating instances\n");
    evm_t::get_cpu_instances(cpu_instances, test);
    printf("%d instances generated\n", cpu_instances.count);

    #ifndef ONLY_CPU
    evm_t::get_gpu_instances(tmp_gpu_instances, cpu_instances);
    CUDA_CHECK(hipMalloc(&gpu_instances, sizeof(evm_instances_t)));
    CUDA_CHECK(hipMemcpy(gpu_instances, &tmp_gpu_instances, sizeof(evm_instances_t), hipMemcpyHostToDevice));
    #endif

    // create a cgbn_error_report for CGBN to report back errors
    #ifndef ONLY_CPU
    size_t heap_size, stack_size;
    CUDA_CHECK(cgbn_error_report_alloc(&report));
    hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
    heap_size = (size_t(2)<<30); // 2GB
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, heap_size));
    // CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 256*1024));
    CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 64*1024));
    printf("Heap size: %zu\n", heap_size);
    hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
    printf("Heap size: %zu\n", heap_size);
    printf("Running GPU kernel ...\n");
    CUDA_CHECK(hipDeviceSynchronize());
    kernel_evm<evm_params><<<cpu_instances.count, evm_params::TPI>>>(report, gpu_instances);
    //CUDA_CHECK(hipPeekAtLastError());
    // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
    CUDA_CHECK(hipDeviceSynchronize());
    printf("GPU kernel finished\n");
    CGBN_CHECK(report);

    // copy the results back to the CPU
    printf("Copying results back to CPU\n");
    CUDA_CHECK(hipMemcpy(&tmp_gpu_instances, gpu_instances, sizeof(evm_instances_t), hipMemcpyDeviceToHost));
    evm_t::get_cpu_instances_from_gpu_instances(cpu_instances, tmp_gpu_instances);
    printf("Results copied\n");
    #else
    printf("Running CPU EVM\n");
    // run the evm
    evm_t *evm = NULL;
    uint32_t tmp_error;
    for(uint32_t instance = 0; instance < cpu_instances.count; instance++) {
      printf("Running instance %d\n", instance);
      evm = new evm_t(
          arith,
          cpu_instances.world_state_data,
          cpu_instances.block_data,
          cpu_instances.sha3_parameters,
          &(cpu_instances.transactions_data[instance]),
          &(cpu_instances.accessed_states_data[instance]),
          &(cpu_instances.touch_states_data[instance]),
          &(cpu_instances.logs_data[instance]),
          #ifdef TRACER
          &(cpu_instances.tracers_data[instance]),
          #endif
          instance,
          &(cpu_instances.errors[instance]));
      evm->run(tmp_error);
      delete evm;
      evm = NULL;
    }
    printf("CPU EVM finished\n");
    #endif


    // print the results
    printf("Printing the results ...\n");
    evm_t::print_evm_instances_t(arith, cpu_instances);
    printf("Results printed\n");

    // print to json files
    printf("Printing to json string ...\n");
    cJSON_AddItemToObject(
      write_root,
      test->string,
      evm_t::json_from_evm_instances_t(arith, cpu_instances));
    printf("Json string printed\n");

    // free the memory
    printf("Freeing the memory ...\n");
    evm_t::free_instances(cpu_instances);
    #ifndef ONLY_CPU
    CUDA_CHECK(hipFree(gpu_instances));
    CUDA_CHECK(cgbn_error_report_free(report));
    CUDA_CHECK(hipDeviceReset());
    #endif
  }
}

void run_json_files(char* read_json_filename, char* write_json_filename){
  //read the json file with the global state
  cJSON *read_root = get_json_from_file(read_json_filename);
  cJSON *write_root = cJSON_CreateObject();

  run_interpreter(read_root, write_root);

  cJSON_Delete(read_root);
  char *json_str=cJSON_Print(write_root);
  FILE *fp=fopen(write_json_filename, "w");
  fprintf(fp, "%s", json_str);
  fclose(fp);
  free(json_str);
  cJSON_Delete(write_root);
}

extern "C" char* run_json_string(const char* read_json_string) {
    cJSON *read_root = cJSON_Parse(read_json_string);
    if (read_root == NULL) {
        // Handle parsing error (optional)
        return NULL;
    }

    cJSON *write_root = cJSON_CreateObject();

    // Assume run_interpreter modifies write_root based on read_root
    run_interpreter(read_root, write_root);
    cJSON_Delete(read_root);
    char *json_str = cJSON_Print(write_root);
    cJSON_Delete(write_root);

    return json_str; // Caller needs to free this memory
}

extern "C" void free_json_string(char* json_str) {
    // temporarily not working (invalid pointer) => potential memory leak
    // in the future, let python manage it with PyObject
    if (json_str) {
      cJSON_free(json_str); // Use the appropriate deallocation function
    }
}


#ifndef BUILD_LIB
int main(int argc, char *argv[]) {//getting the input
  char *read_json_filename = NULL;
  char *write_json_filename = NULL;
  static struct option long_options[] = {
        {"input", required_argument, 0, 'i'},
        {"output", required_argument, 0, 'o'},
        {0, 0, 0, 0}};

  int opt;
  int option_index = 0;
  while ((opt = getopt_long(argc, argv, "i:o:", long_options, &option_index)) != -1)
  {
      switch (opt)
      {
      case 'i':
          read_json_filename = optarg;
          break;
      case 'o':
          write_json_filename = optarg;
          break;
      default:
          fprintf(stderr, "Usage: %s --input <json_filename> --output <json_filename>\n", argv[0]);
          exit(EXIT_FAILURE);
      }
  }
  if (!read_json_filename || !write_json_filename)
  {
      fprintf(stderr, "Both --input and --output flags are required\n");
      exit(EXIT_FAILURE);
  }

  // check if the file exists
  std::ifstream file(read_json_filename);
  if (!file) {
    fprintf(stderr, "File '%s' does not exist\n", read_json_filename);
    exit(EXIT_FAILURE);
  }
  run_json_files(read_json_filename, write_json_filename);
}
#endif
