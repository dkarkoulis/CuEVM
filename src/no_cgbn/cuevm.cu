#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <getopt.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuevm_test.h"
#include "opcode.h"
#include "processor.cuh"
#include "stack.cuh"
#include "uint256.cuh"
#define NUMTHREAD 4096
#define DEBUG 1
// simple draft kernel for place holder
// simple testing opcodes and return the popped top of stack value
__global__ void cuEVM(unsigned char *bytecode, unsigned char *input, size_t bytecode_len, size_t input_len,
                      size_t num_threads) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < num_threads) {
        if (idx == 0) {
            printf("Bytecode: ");
            for (size_t i = 0; i < bytecode_len; i++) {
                printf("%02x ", bytecode[i]);
            }
            printf("\n");

            printf("Input: ");
            for (size_t i = 0; i < input_len; i++) {
                printf("%02x ", input[i]);
            }
            printf("\n");

            // todo refactor processor.execute to a function outside of this kernel
            processor evm;
            evm.programCounter = 0;  // redundant but better safe.
            init_stack(&evm.stack);

            // push(&stack, a);
            // pop(&stack, &b);
            // debugging : print_stack(&stack);
            // define 3 reusable temp uints for binary op
            base_uint op1, op2, result;
            uint safe_counter = 0;  // safety counter prevent infinite loop
            while (evm.programCounter < bytecode_len) {
                unsigned char opcode = bytecode[evm.programCounter];
                safe_counter++;
                if (safe_counter > 100) {
                    printf("Safety counter exceeded, return from execution\n");
                    return;
                }
                switch (opcode) {
                    case ADD:  // ADD
                        // TODO: check stack size
                        // future optimization : can override push pop ops and modify the stack directly
                        pop(&evm.stack, &op1);
                        pop(&evm.stack, &op2);
                        base_uint_add(&op1, &op2, &result);

#if DEBUG
                        printf("ADD OPCODE: \n");
                        printf("op1: ");
                        print_base_uint(&op1);
                        printf("op2: ");
                        print_base_uint(&op2);
                        printf("result: ");
                        print_base_uint(&result);

#endif

                        push(&evm.stack, result);
                        break;
                    case SUB:
                        pop(&evm.stack, &op1);
                        pop(&evm.stack, &op2);
                        base_uint_sub(&op1, &op2, &result);

#if DEBUG
                        printf("SUB OPCODE: \n");
                        printf("op1: ");
                        print_base_uint(&op1);
                        printf("op2: ");
                        print_base_uint(&op2);
                        printf("result: ");
                        print_base_uint(&result);

#endif

                        push(&evm.stack, result);
                        break;

                    case MUL:  // MUL
                        // TODO: check stack size
                        pop(&evm.stack, &op1);
                        pop(&evm.stack, &op2);
                        base_uint_mul(&op1, &op2, &result);

#if DEBUG
                        printf("MUL OPCODE: \n");
                        printf("op1: ");
                        print_base_uint(&op1);
                        printf("op2: ");
                        print_base_uint(&op2);
                        printf("result: ");
                        print_base_uint(&result);

#endif

                        push(&evm.stack, result);
                        break;
                    case PUSH1:
                        unsigned char push_val = bytecode[++evm.programCounter];
                        result = {{push_val, 0, 0, 0, 0, 0, 0, 0}};
                        push(&evm.stack, result);

#if DEBUG
                        printf("PUSH1 OPCODE: \n");
                        printf("push_val: ");
                        print_base_uint(&result);

#endif

                        break;
                    case PUSH2:
                        // Increment the program counter to point to the first byte of data
                        evm.programCounter++;

                        // Read the two bytes from the bytecode
                        unsigned char byte1 = bytecode[evm.programCounter];
                        unsigned char byte2 = bytecode[++evm.programCounter];

                        // Combine the two bytes into a single 16-bit value
                        uint16_t push_val_16 = (byte1 << 8) | byte2;

                        // Convert the 16-bit value into your base_uint format
                        result = {{push_val_16, 0, 0, 0, 0, 0, 0, 0}};

                        // Push the value onto the stack
                        push(&evm.stack, result);

#if DEBUG
                        printf("PUSH2 OPCODE: \n");
                        printf("push_val: ");
                        print_base_uint(&result);

#endif

                        break;

                    case POP:
                        pop(&evm.stack, &result);
                        printf("Popped Stack value: ");
                        print_base_uint(&result);
                        break;

                    case SWAP1:
                        pop(&evm.stack, &op1);
                        pop(&evm.stack, &op2);
                        push(&evm.stack, op1);
                        push(&evm.stack, op2);
                        break;

                    case DUP1:
                        printf("DUP1 OPCODE: \n");
                        push(&evm.stack, evm.stack.items[evm.stack.top]);
                        break;
                    case JUMPI:
                        pop(&evm.stack, &op1);
                        pop(&evm.stack, &op2);
#if DEBUG
                        printf("JUMPI OPCODE: \n");
                        printf("Condition:\n");
                        print_base_uint(&op2);
                        printf("Destination:\n");
                        print_base_uint(&op1);
                        printf("is ZEROP op1: %d\n", is_zero(&op1));
#endif
                        if (!is_zero(&op2)) {
                            evm.programCounter = op1.pn[0];
                            // TODO: check JUMPDEST in destiation ?
                        }
                        break;

                    case JUMP:
                        pop(&evm.stack, &op1);
                        evm.programCounter = op1.pn[0];
                        // TODO: check JUMPDEST in destiation ?
                        break;

                    case JUMPDEST:
                        // do nothing
                        break;
                    case RETURN:
                        printf("RETURN OPCODE\n");
                        evm.programCounter = bytecode_len;
                        break;
                    default:
                        printf("Unknown opcode %d at position %d\n", opcode, evm.programCounter);
                        printf("Return from execution\n");
                        return;
                }

                evm.programCounter++;
#if DEBUG
                printf("Program counter: %d\n", evm.programCounter);
                printf("Stack size: %d\n", evm.stack.top + 1);
                print_stack(&evm.stack);
                printf("\n***************\n");
#endif
            }
        }
    }
}
int adjustedLength(char **hexString) {
    if (strncmp(*hexString, "0x", 2) == 0 || strncmp(*hexString, "0X", 2) == 0) {
        *hexString += 2;  // Skip the "0x" prefix
        return (strlen(*hexString) / 2);
    }
    return (strlen(*hexString) / 2);
}

void hexStringToByteArray(const char *hexString, unsigned char *byteArray, int length) {
    for (int i = 0; i < length; i += 2) {
        sscanf(&hexString[i], "%2hhx", &byteArray[i / 2]);
    }
}

int main(int argc, char *argv[]) {
    char *byte_code_hex = NULL;
    char *input_hex = NULL;

    static struct option long_options[] = {{"bytecode", required_argument, 0, 'b'},
                                           {"input", required_argument, 0, 'i'},
                                           {"test", no_argument, 0, 't'},
                                           {0, 0, 0, 0}};

    int opt;
    int option_index = 0;
    while ((opt = getopt_long(argc, argv, "b:i:", long_options, &option_index)) != -1) {
        switch (opt) {
            case 'b':
                byte_code_hex = optarg;
                break;
            case 'i':
                input_hex = optarg;
                break;
            case 't':
                test_arithmetic_operations();
                test_stack();
                exit(0);
            default:
                fprintf(stdout, "Usage: %s --bytecode <hexstring> --input <hexstring>\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    if (!byte_code_hex || !input_hex) {
        fprintf(stdout, "Both --bytecode and --input flags are required\n");
        exit(EXIT_FAILURE);
    }

    int bytecode_len = adjustedLength(&byte_code_hex);
    int input_len = adjustedLength(&input_hex);

    unsigned char *byte_code = (unsigned char *)malloc(bytecode_len);
    unsigned char *input = (unsigned char *)malloc(input_len);

    hexStringToByteArray(byte_code_hex, byte_code, bytecode_len * 2);
    hexStringToByteArray(input_hex, input, input_len * 2);

    unsigned char *d_bytecode, *d_input;
    hipMalloc((void **)&d_bytecode, bytecode_len);
    hipMalloc((void **)&d_input, input_len);

    hipMemcpy(d_bytecode, byte_code, bytecode_len, hipMemcpyHostToDevice);
    hipMemcpy(d_input, input, input_len, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (NUMTHREAD + blockSize - 1) / blockSize;
    cuEVM<<<numBlocks, blockSize>>>(d_bytecode, d_input, bytecode_len, input_len, NUMTHREAD);

    hipDeviceSynchronize();

    hipFree(d_bytecode);
    hipFree(d_input);
    free(byte_code);
    free(input);

    return 0;
}
