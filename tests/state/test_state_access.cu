#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <CuEVM/state/account.cuh>
#include <CuEVM/state/account_flags.cuh>
#include <CuEVM/state/state_access.cuh>
#include <CuEVM/utils/arith.cuh>
#include <CuEVM/utils/error_codes.cuh>

using namespace CuEVM;

class StateAccessTest : public ::testing::Test {
   protected:
    CuEVM::ArithEnv arith;
    CuEVM::state_access_t empty_state;
    CuEVM::state_access_t filled_state;
    CuEVM::account_t account;

    StateAccessTest() : arith(cgbn_no_checks), empty_state(), filled_state(), account() {
        CuEVM::bn_t value;
        cgbn_set_ui32(arith.env, value, 0x12345678);
        account.set_address(arith, value);
        cgbn_set_ui32(arith.env, value, 0x1000);
        account.set_balance(arith, value);
        cgbn_set_ui32(arith.env, value, 0x1);
        account.set_nonce(arith, value);
        CuEVM::byte_array_t byte_code(3);
        byte_code[0] = 0xFF;
        byte_code[1] = 0x00;
        byte_code[2] = 0x11;
        account.set_byte_code(byte_code);
        CuEVM::bn_t key;
        cgbn_set_ui32(arith.env, key, 0x1);
        cgbn_set_ui32(arith.env, value, 0x2);
        account.set_storage_value(arith, key, value);
        filled_state.set_account(arith, account, CuEVM::ACCOUNT_NONE_FLAG);
    }

    void SetUp() override {
        // Initialize any necessary resources here
    }

    void TearDown() override {
        // Clean up any resources here
        empty_state.free();
        account.free();
    }
};

TEST_F(StateAccessTest, DefaultConstructor) {
    EXPECT_EQ(empty_state.no_accounts, 0U);
    EXPECT_EQ(empty_state.accounts, nullptr);
    EXPECT_EQ(empty_state.flags, nullptr);
}

TEST_F(StateAccessTest, CopyConstructor) {
    state_access_t state2(filled_state);
    EXPECT_EQ(state2.no_accounts, filled_state.no_accounts);
    EXPECT_EQ(state2.accounts[0].address, filled_state.accounts[0].address);
    EXPECT_EQ(state2.accounts[0].balance, filled_state.accounts[0].balance);
    EXPECT_EQ(state2.accounts[0].nonce, filled_state.accounts[0].nonce);
    EXPECT_EQ(state2.accounts[0].byte_code.size, filled_state.accounts[0].byte_code.size);
    EXPECT_EQ(state2.accounts[0].byte_code[0], filled_state.accounts[0].byte_code[0]);
    EXPECT_EQ(state2.accounts[0].byte_code[1], filled_state.accounts[0].byte_code[1]);
    EXPECT_EQ(state2.accounts[0].byte_code[2], filled_state.accounts[0].byte_code[2]);
    EXPECT_EQ(state2.accounts[0].storage.size, filled_state.accounts[0].storage.size);
    EXPECT_EQ(state2.accounts[0].storage.capacity, filled_state.accounts[0].storage.capacity);
    EXPECT_EQ(state2.accounts[0].storage.storage[0].key, filled_state.accounts[0].storage.storage[0].key);
    EXPECT_EQ(state2.accounts[0].storage.storage[0].value, filled_state.accounts[0].storage.storage[0].value);
    EXPECT_EQ(state2.flags[0].flags, filled_state.flags[0].flags);
}

TEST_F(StateAccessTest, AssignmentOperator) {
    state_access_t state2;
    state2 = filled_state;
    EXPECT_EQ(state2.no_accounts, filled_state.no_accounts);
    EXPECT_EQ(state2.accounts[0].address, filled_state.accounts[0].address);
    EXPECT_EQ(state2.accounts[0].balance, filled_state.accounts[0].balance);
    EXPECT_EQ(state2.accounts[0].nonce, filled_state.accounts[0].nonce);
    EXPECT_EQ(state2.accounts[0].byte_code.size, filled_state.accounts[0].byte_code.size);
    EXPECT_EQ(state2.accounts[0].byte_code[0], filled_state.accounts[0].byte_code[0]);
    EXPECT_EQ(state2.accounts[0].byte_code[1], filled_state.accounts[0].byte_code[1]);
    EXPECT_EQ(state2.accounts[0].byte_code[2], filled_state.accounts[0].byte_code[2]);
    EXPECT_EQ(state2.accounts[0].storage.size, filled_state.accounts[0].storage.size);
    EXPECT_EQ(state2.accounts[0].storage.capacity, filled_state.accounts[0].storage.capacity);
    EXPECT_EQ(state2.accounts[0].storage.storage[0].key, filled_state.accounts[0].storage.storage[0].key);
    EXPECT_EQ(state2.accounts[0].storage.storage[0].value, filled_state.accounts[0].storage.storage[0].value);
    EXPECT_EQ(state2.flags[0].flags, filled_state.flags[0].flags);
}

TEST_F(StateAccessTest, AddAccount) {
    EXPECT_EQ(empty_state.add_account(account), ERROR_SUCCESS);
    EXPECT_EQ(empty_state.no_accounts, 1U);
    EXPECT_EQ(empty_state.accounts[0].address, account.address);
    EXPECT_EQ(empty_state.accounts[0].balance, account.balance);
    EXPECT_EQ(empty_state.accounts[0].nonce, account.nonce);
    EXPECT_EQ(empty_state.accounts[0].byte_code.size, account.byte_code.size);
    EXPECT_EQ(empty_state.accounts[0].byte_code[0], account.byte_code[0]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[1], account.byte_code[1]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[2], account.byte_code[2]);
    EXPECT_EQ(empty_state.accounts[0].storage.size, account.storage.size);
    EXPECT_EQ(empty_state.accounts[0].storage.capacity, account.storage.capacity);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].value, account.storage.storage[0].value);
    EXPECT_EQ(empty_state.flags[0].flags, CuEVM::ACCOUNT_NONE_FLAG);
}

TEST_F(StateAccessTest, SetAccount) {
    empty_state.set_account(arith, account);
    EXPECT_EQ(empty_state.no_accounts, 1U);
    EXPECT_EQ(empty_state.accounts[0].address, account.address);
    EXPECT_EQ(empty_state.accounts[0].balance, account.balance);
    EXPECT_EQ(empty_state.accounts[0].nonce, account.nonce);
    EXPECT_EQ(empty_state.accounts[0].byte_code.size, account.byte_code.size);
    EXPECT_EQ(empty_state.accounts[0].byte_code[0], account.byte_code[0]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[1], account.byte_code[1]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[2], account.byte_code[2]);
    EXPECT_EQ(empty_state.accounts[0].storage.size, account.storage.size);
    EXPECT_EQ(empty_state.accounts[0].storage.capacity, account.storage.capacity);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].value, account.storage.storage[0].value);
    EXPECT_EQ(empty_state.flags[0].flags, CuEVM::ACCOUNT_ALL_FLAG);
}

TEST_F(StateAccessTest, GetAccount) {
    CuEVM::account_t retrieved_account;
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, account.address);
    EXPECT_EQ(retrieved_account.balance, account.balance);
    EXPECT_EQ(retrieved_account.nonce, account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], account.byte_code[1]);
    EXPECT_EQ(retrieved_account.byte_code[2], account.byte_code[2]);
    EXPECT_EQ(retrieved_account.storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, account.storage.storage[0].value);

    CuEVM::bn_t non_existent_address;
    cgbn_set_ui32(arith.env, non_existent_address, 0x87654321);
    EXPECT_EQ(filled_state.get_account(arith, non_existent_address, retrieved_account), ERROR_STATE_ADDRESS_NOT_FOUND);

    // Test get account ptr
    CuEVM::account_t* retrieved_account_ptr;
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account_ptr), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account_ptr->address, account.address);
    EXPECT_EQ(retrieved_account_ptr->balance, account.balance);
    EXPECT_EQ(retrieved_account_ptr->nonce, account.nonce);
    EXPECT_EQ(retrieved_account_ptr->byte_code.size, account.byte_code.size);
    EXPECT_EQ(retrieved_account_ptr->byte_code[0], account.byte_code[0]);
    EXPECT_EQ(retrieved_account_ptr->byte_code[1], account.byte_code[1]);
    EXPECT_EQ(retrieved_account_ptr->byte_code[2], account.byte_code[2]);
    EXPECT_EQ(retrieved_account_ptr->storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account_ptr->storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account_ptr->storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account_ptr->storage.storage[0].value, account.storage.storage[0].value);
}

TEST_F(StateAccessTest, GetAccountWithFlags) {
    empty_state.set_account(arith, account, CuEVM::ACCOUNT_NONE_FLAG);
    EXPECT_EQ(empty_state.no_accounts, 1U);
    EXPECT_EQ(empty_state.accounts[0].address, account.address);
    EXPECT_EQ(empty_state.accounts[0].balance, account.balance);
    EXPECT_EQ(empty_state.accounts[0].nonce, account.nonce);
    EXPECT_EQ(empty_state.accounts[0].byte_code.size, account.byte_code.size);
    EXPECT_EQ(empty_state.accounts[0].byte_code[0], account.byte_code[0]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[1], account.byte_code[1]);
    EXPECT_EQ(empty_state.accounts[0].byte_code[2], account.byte_code[2]);
    EXPECT_EQ(empty_state.accounts[0].storage.size, account.storage.size);
    EXPECT_EQ(empty_state.accounts[0].storage.capacity, account.storage.capacity);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(empty_state.accounts[0].storage.storage[0].value, account.storage.storage[0].value);
    EXPECT_EQ(empty_state.flags[0].flags, CuEVM::ACCOUNT_NONE_FLAG);
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    CuEVM::account_t* retrieved_account_ptr;
    EXPECT_EQ(empty_state.get_account(arith, address, retrieved_account_ptr, CuEVM::ACCOUNT_STORAGE_FLAG),
              ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account_ptr->address, account.address);
    EXPECT_EQ(retrieved_account_ptr->balance, account.balance);
    EXPECT_EQ(retrieved_account_ptr->nonce, account.nonce);
    EXPECT_EQ(retrieved_account_ptr->byte_code.size, account.byte_code.size);
    EXPECT_EQ(retrieved_account_ptr->byte_code[0], account.byte_code[0]);
    EXPECT_EQ(retrieved_account_ptr->byte_code[1], account.byte_code[1]);
    EXPECT_EQ(retrieved_account_ptr->byte_code[2], account.byte_code[2]);
    EXPECT_EQ(retrieved_account_ptr->storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account_ptr->storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account_ptr->storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account_ptr->storage.storage[0].value, account.storage.storage[0].value);
    EXPECT_EQ(empty_state.flags[0].flags, CuEVM::ACCOUNT_STORAGE_FLAG);
}

TEST_F(StateAccessTest, HasAccount) {
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    EXPECT_EQ(filled_state.has_account(arith, address), ERROR_SUCCESS);

    CuEVM::bn_t non_existent_address;
    cgbn_set_ui32(arith.env, non_existent_address, 0x87654321);
    EXPECT_EQ(filled_state.has_account(arith, non_existent_address), ERROR_STATE_ADDRESS_NOT_FOUND);
}

TEST_F(StateAccessTest, UpdateAccount) {
    CuEVM::account_t updated_account;
    updated_account.address = account.address;
    updated_account.balance = 100U;
    updated_account.nonce = 2U;
    updated_account.byte_code.clear();
    updated_account.byte_code.grow(2, 1);
    updated_account.byte_code[0] = 0x11;
    updated_account.byte_code[1] = 0x22;
    updated_account.storage.clear();
    CuEVM::bn_t key, value;
    cgbn_set_ui32(arith.env, key, 0x1);
    cgbn_set_ui32(arith.env, value, 0x3);
    updated_account.set_storage_value(arith, key, value);
    cgbn_set_ui32(arith.env, key, 0x2);
    cgbn_set_ui32(arith.env, value, 0x4);
    updated_account.set_storage_value(arith, key, value);
    EXPECT_EQ(filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_NONCE_FLAG), ERROR_SUCCESS);

    CuEVM::account_t retrieved_account;
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, account.address);
    EXPECT_EQ(retrieved_account.balance, account.balance);
    EXPECT_EQ(retrieved_account.nonce, updated_account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], account.byte_code[1]);
    EXPECT_EQ(retrieved_account.byte_code[2], account.byte_code[2]);
    EXPECT_EQ(retrieved_account.storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, account.storage.storage[0].value);

    EXPECT_EQ(filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_BALANCE_FLAG), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, account.address);
    EXPECT_EQ(retrieved_account.balance, updated_account.balance);
    EXPECT_EQ(retrieved_account.nonce, updated_account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], account.byte_code[1]);
    EXPECT_EQ(retrieved_account.byte_code[2], account.byte_code[2]);
    EXPECT_EQ(retrieved_account.storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, account.storage.storage[0].value);

    EXPECT_EQ(filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_BYTE_CODE_FLAG), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, account.address);
    EXPECT_EQ(retrieved_account.balance, updated_account.balance);
    EXPECT_EQ(retrieved_account.nonce, updated_account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, updated_account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], updated_account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], updated_account.byte_code[1]);
    EXPECT_EQ(retrieved_account.storage.size, account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, account.storage.storage[0].value);

    EXPECT_EQ(filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_STORAGE_FLAG), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, account.address);
    EXPECT_EQ(retrieved_account.balance, updated_account.balance);
    EXPECT_EQ(retrieved_account.nonce, updated_account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, updated_account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], updated_account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], updated_account.byte_code[1]);
    EXPECT_EQ(retrieved_account.storage.size, updated_account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, updated_account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, updated_account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, updated_account.storage.storage[0].value);
    EXPECT_EQ(retrieved_account.storage.storage[1].key, updated_account.storage.storage[1].key);
    EXPECT_EQ(retrieved_account.storage.storage[1].value, updated_account.storage.storage[1].value);
}

TEST_F(StateAccessTest, IsEmptyAccount) {
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    EXPECT_EQ(filled_state.has_account(arith, address), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.is_empty_account(arith, address), ERROR_ACCOUNT_NOT_EMPTY);
    cgbn_set_ui32(arith.env, address, 0x87654321);
    EXPECT_EQ(filled_state.has_account(arith, address), ERROR_STATE_ADDRESS_NOT_FOUND);
    EXPECT_EQ(filled_state.is_empty_account(arith, address), ERROR_STATE_ADDRESS_NOT_FOUND);
    CuEVM::account_t empty_account;
    empty_account.empty();
    cgbn_store(arith.env, &(empty_account.address), address);
    EXPECT_EQ(filled_state.set_account(arith, empty_account), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.has_account(arith, address), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.is_empty_account(arith, address), ERROR_SUCCESS);
}

TEST_F(StateAccessTest, UpdateState) {
    CuEVM::state_access_t updated_state;
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x87654321);
    CuEVM::account_t updated_account;
    updated_account.set_address(arith, address);
    updated_account.balance = 100U;
    updated_account.nonce = 2U;
    updated_account.byte_code.clear();
    updated_account.byte_code.grow(2, 1);
    updated_account.byte_code[0] = 0x11;
    updated_account.byte_code[1] = 0x22;
    updated_account.storage.clear();
    CuEVM::bn_t key, value;
    cgbn_set_ui32(arith.env, key, 0x1);
    cgbn_set_ui32(arith.env, value, 0x3);
    updated_account.set_storage_value(arith, key, value);
    cgbn_set_ui32(arith.env, key, 0x2);
    cgbn_set_ui32(arith.env, value, 0x4);
    updated_account.set_storage_value(arith, key, value);
    updated_state.set_account(arith, updated_account, CuEVM::ACCOUNT_ALL_FLAG);
    EXPECT_EQ(filled_state.update(arith, updated_state), ERROR_SUCCESS);
    EXPECT_EQ(filled_state.no_accounts, 2U);

    CuEVM::account_t retrieved_account;
    cgbn_set_ui32(arith.env, address, 0x87654321);
    EXPECT_EQ(filled_state.get_account(arith, address, retrieved_account), ERROR_SUCCESS);
    EXPECT_EQ(retrieved_account.address, updated_account.address);
    EXPECT_EQ(retrieved_account.balance, updated_account.balance);
    EXPECT_EQ(retrieved_account.nonce, updated_account.nonce);
    EXPECT_EQ(retrieved_account.byte_code.size, updated_account.byte_code.size);
    EXPECT_EQ(retrieved_account.byte_code[0], updated_account.byte_code[0]);
    EXPECT_EQ(retrieved_account.byte_code[1], updated_account.byte_code[1]);
    EXPECT_EQ(retrieved_account.storage.size, updated_account.storage.size);
    EXPECT_EQ(retrieved_account.storage.capacity, updated_account.storage.capacity);
    EXPECT_EQ(retrieved_account.storage.storage[0].key, updated_account.storage.storage[0].key);
    EXPECT_EQ(retrieved_account.storage.storage[0].value, updated_account.storage.storage[0].value);
    EXPECT_EQ(retrieved_account.storage.storage[1].key, updated_account.storage.storage[1].key);
    EXPECT_EQ(retrieved_account.storage.storage[1].value, updated_account.storage.storage[1].value);
    EXPECT_EQ(filled_state.flags[1].flags, CuEVM::ACCOUNT_ALL_FLAG);
}

TEST_F(StateAccessTest, ToJson) {
    cJSON* json = filled_state.to_json();
    char* json_str = cJSON_Print(json);
    EXPECT_STREQ(
        json_str,
        "{\n\t\"0x0000000000000000000000000000000012345678\":\t{\n\t\t\"balance\":"
        "\t\"0x0000000000000000000000000000000000000000000000000000000000001000\",\n\t\t\"nonce\":"
        "\t\"0x0000000000000000000000000000000000000000000000000000000000000001\",\n\t\t\"code\":\t\"0xff0011\","
        "\n\t\t\"storage\":\t{\n\t\t\t\"0x1\":\t\"0x2\"\n\t\t},\n\t\t\"flags\":\t\"00000000\"\n\t}\n}");
    cJSON_Delete(json);
    free(json_str);
}

TEST_F(StateAccessTest, Print) {
    // Redirect stdout to a string stream
    testing::internal::CaptureStdout();

    filled_state.print();
    std::string output = testing::internal::GetCapturedStdout();

    // Expected output
    ASSERT_EQ(output,
              "no_accounts: 1\naccounts[0]:\nAccount:\n00000000 00000000 00000000 00000000 00000000 00000000 00000000 "
              "12345678 \nBalance: 00000000 00000000 00000000 00000000 00000000 00000000 00000000 00001000 \nNonce: "
              "00000000 00000000 00000000 00000000 00000000 00000000 00000000 00000001 \nByte code: size: 3\ndata: "
              "ff0011\nStorage: \nStorage size: 1\nElement 0:\nKey: 00000000 00000000 00000000 00000000 00000000 "
              "00000000 00000000 00000001 \nValue: 00000000 00000000 00000000 00000000 00000000 00000000 00000000 "
              "00000002 \nflags[0]:\nAccount flags: 00000000\n");
}

__global__ void test_state_access_kernel(CuEVM::state_access_t* gpu_states, uint32_t* result, uint32_t count) {
    int32_t instance = (blockIdx.x * blockDim.x + threadIdx.x) / CuEVM::cgbn_tpi;
    if (instance >= count) return;
    result[instance] = ERROR_SUCCESS;
    CuEVM::ArithEnv arith(cgbn_no_checks);

    CuEVM::state_access_t empty_state;
    CuEVM::state_access_t filled_state;
    CuEVM::account_t account;
    CuEVM::bn_t value;
    cgbn_set_ui32(arith.env, value, 0x12345678);
    account.set_address(arith, value);
    cgbn_set_ui32(arith.env, value, 0x1000);
    account.set_balance(arith, value);
    cgbn_set_ui32(arith.env, value, 0x1);
    account.set_nonce(arith, value);
    CuEVM::byte_array_t byte_code(3);
    byte_code[0] = 0xFF;
    byte_code[1] = 0x00;
    byte_code[2] = 0x11;
    account.set_byte_code(byte_code);
    CuEVM::bn_t key;
    cgbn_set_ui32(arith.env, key, 0x1);
    cgbn_set_ui32(arith.env, value, 0x2);
    account.set_storage_value(arith, key, value);

    filled_state.set_account(arith, account, CuEVM::ACCOUNT_NONE_FLAG);
    // return;

    // Test Default Constructor
    result[instance] = (empty_state.no_accounts == 0U) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts == nullptr) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.flags == nullptr) ? ERROR_SUCCESS : __LINE__;

    // Test Copy Constructor
    CuEVM::state_access_t state2(filled_state);
    result[instance] |= (state2.no_accounts == filled_state.no_accounts) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state2.accounts[0].address == filled_state.accounts[0].address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state2.accounts[0].balance == filled_state.accounts[0].balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state2.accounts[0].nonce == filled_state.accounts[0].nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].byte_code.size == filled_state.accounts[0].byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].byte_code[0] == filled_state.accounts[0].byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].byte_code[1] == filled_state.accounts[0].byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].byte_code[2] == filled_state.accounts[0].byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].storage.size == filled_state.accounts[0].storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state2.accounts[0].storage.capacity == filled_state.accounts[0].storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state2.accounts[0].storage.storage[0].key == filled_state.accounts[0].storage.storage[0].key)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |=
        (state2.accounts[0].storage.storage[0].value == filled_state.accounts[0].storage.storage[0].value)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |= (state2.flags[0].flags == filled_state.flags[0].flags) ? ERROR_SUCCESS : __LINE__;

    // Test Assignment Operator
    CuEVM::state_access_t state3;
    state3 = filled_state;
    result[instance] |= (state3.no_accounts == filled_state.no_accounts) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state3.accounts[0].address == filled_state.accounts[0].address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state3.accounts[0].balance == filled_state.accounts[0].balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state3.accounts[0].nonce == filled_state.accounts[0].nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].byte_code.size == filled_state.accounts[0].byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].byte_code[0] == filled_state.accounts[0].byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].byte_code[1] == filled_state.accounts[0].byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].byte_code[2] == filled_state.accounts[0].byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].storage.size == filled_state.accounts[0].storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (state3.accounts[0].storage.capacity == filled_state.accounts[0].storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (state3.accounts[0].storage.storage[0].key == filled_state.accounts[0].storage.storage[0].key)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |=
        (state3.accounts[0].storage.storage[0].value == filled_state.accounts[0].storage.storage[0].value)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |= (state3.flags[0].flags == filled_state.flags[0].flags) ? ERROR_SUCCESS : __LINE__;

    // Test Add Account
    result[instance] |= (empty_state.add_account(account) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.no_accounts == 1U) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (empty_state.accounts[0].storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (empty_state.accounts[0].storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].storage.storage[0].value == account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (empty_state.flags[0].flags == CuEVM::ACCOUNT_NONE_FLAG) ? ERROR_SUCCESS : __LINE__;

    empty_state.free();

    // Test Set Account
    empty_state.set_account(arith, account);
    result[instance] |= (empty_state.no_accounts == 1U) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (empty_state.accounts[0].storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (empty_state.accounts[0].storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].storage.storage[0].value == account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (empty_state.flags[0].flags == CuEVM::ACCOUNT_ALL_FLAG) ? ERROR_SUCCESS : __LINE__;

    // Test Get Account
    CuEVM::account_t retrieved_account;
    CuEVM::bn_t address;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account.storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account.storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account.storage.storage[0].value == account.storage.storage[0].value) ? ERROR_SUCCESS : __LINE__;
    CuEVM::bn_t non_existent_address;
    cgbn_set_ui32(arith.env, non_existent_address, 0x87654321);
    result[instance] |=
        (filled_state.get_account(arith, non_existent_address, retrieved_account) == ERROR_STATE_ADDRESS_NOT_FOUND)
            ? ERROR_SUCCESS
            : __LINE__;
    CuEVM::account_t* retrieved_account_ptr;
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account_ptr) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account_ptr->storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account_ptr->storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->storage.storage[0].value == account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    // Test Get Account with Flags
    empty_state.free();
    empty_state.set_account(arith, account, CuEVM::ACCOUNT_NONE_FLAG);
    result[instance] |= (empty_state.no_accounts == 1U) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.accounts[0].storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (empty_state.accounts[0].storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.flags[0].flags == CuEVM::ACCOUNT_NONE_FLAG) ? ERROR_SUCCESS : __LINE__;

    cgbn_set_ui32(arith.env, address, 0x12345678);
    result[instance] |=
        (empty_state.get_account(arith, address, retrieved_account_ptr, CuEVM::ACCOUNT_STORAGE_FLAG) == ERROR_SUCCESS)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |= (retrieved_account_ptr->address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->nonce == account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account_ptr->storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account_ptr->storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account_ptr->storage.storage[0].value == account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (empty_state.flags[0].flags == CuEVM::ACCOUNT_STORAGE_FLAG) ? ERROR_SUCCESS : __LINE__;

    // Test Has Account
    result[instance] |= (filled_state.has_account(arith, address) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.has_account(arith, non_existent_address) == ERROR_STATE_ADDRESS_NOT_FOUND)
                            ? ERROR_SUCCESS
                            : __LINE__;

    // Test Update Account
    CuEVM::account_t updated_account;
    updated_account.address = account.address;
    updated_account.balance = 0x100U;
    updated_account.nonce = 0x2U;
    updated_account.byte_code.clear();
    updated_account.byte_code.grow(2, 1);
    updated_account.byte_code[0] = 0x11;
    updated_account.byte_code[1] = 0x22;
    updated_account.storage.clear();
    cgbn_set_ui32(arith.env, key, 0x1);
    cgbn_set_ui32(arith.env, value, 0x3);
    updated_account.set_storage_value(arith, key, value);
    cgbn_set_ui32(arith.env, key, 0x2);
    cgbn_set_ui32(arith.env, value, 0x4);
    updated_account.set_storage_value(arith, key, value);
    result[instance] |=
        (filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_NONCE_FLAG) == ERROR_SUCCESS)
            ? ERROR_SUCCESS
            : __LINE__;
    CuEVM::account_t retrieved_account2;
    cgbn_set_ui32(arith.env, address, 0x12345678);
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account2) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.balance == account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.nonce == updated_account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].value == account.storage.storage[0].value) ? ERROR_SUCCESS : __LINE__;

    result[instance] |=
        (filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_BALANCE_FLAG) == ERROR_SUCCESS)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account2) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.balance == updated_account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.nonce == updated_account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code.size == account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[0] == account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[1] == account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[2] == account.byte_code[2]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].value == account.storage.storage[0].value) ? ERROR_SUCCESS : __LINE__;

    result[instance] |=
        (filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_BYTE_CODE_FLAG) == ERROR_SUCCESS)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account2) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.balance == updated_account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.nonce == updated_account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.byte_code.size == updated_account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[0] == updated_account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[1] == updated_account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.size == account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.capacity == account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].key == account.storage.storage[0].key) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.storage[0].value == account.storage.storage[0].value) ? ERROR_SUCCESS : __LINE__;

    result[instance] |=
        (filled_state.update_account(arith, updated_account, CuEVM::ACCOUNT_STORAGE_FLAG) == ERROR_SUCCESS)
            ? ERROR_SUCCESS
            : __LINE__;
    result[instance] |=
        (filled_state.get_account(arith, address, retrieved_account2) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.address == account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.balance == updated_account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.nonce == updated_account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.byte_code.size == updated_account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[0] == updated_account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.byte_code[1] == updated_account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.size == updated_account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (retrieved_account2.storage.capacity == updated_account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (retrieved_account2.storage.storage[0].key == updated_account.storage.storage[0].key)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (retrieved_account2.storage.storage[0].value == updated_account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (retrieved_account2.storage.storage[1].key == updated_account.storage.storage[1].key)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (retrieved_account2.storage.storage[1].value == updated_account.storage.storage[1].value)
                            ? ERROR_SUCCESS
                            : __LINE__;

    // Test Is Empty Account
    result[instance] |=
        (filled_state.is_empty_account(arith, address) == ERROR_ACCOUNT_NOT_EMPTY) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.is_empty_account(arith, non_existent_address) == ERROR_STATE_ADDRESS_NOT_FOUND)
                            ? ERROR_SUCCESS
                            : __LINE__;
    CuEVM::account_t empty_account;
    empty_account.empty();
    cgbn_store(arith.env, &(empty_account.address), address);
    result[instance] |= (empty_state.set_account(arith, empty_account) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.has_account(arith, address) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (empty_state.is_empty_account(arith, address) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;

    // add the empty state to the filled state
    empty_account.set_address(arith, non_existent_address);
    result[instance] |= (filled_state.set_account(arith, empty_account) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (filled_state.has_account(arith, non_existent_address) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (filled_state.is_empty_account(arith, non_existent_address) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;

    // add a new state to the filled state
    CuEVM::account_t new_account;
    new_account.clear();
    cgbn_set_ui32(arith.env, value, 0x87651234);
    new_account.set_address(arith, value);
    cgbn_set_ui32(arith.env, value, 0);
    new_account.set_balance(arith, value);
    cgbn_set_ui32(arith.env, value, 0);
    new_account.set_nonce(arith, value);
    new_account.set_byte_code(byte_code);
    cgbn_set_ui32(arith.env, key, 0x1);
    cgbn_set_ui32(arith.env, value, 0x3);
    new_account.set_storage_value(arith, key, value);
    CuEVM::state_access_t new_state;
    new_state.clear();
    new_state.set_account(arith, new_account, CuEVM::ACCOUNT_ALL_FLAG);
    result[instance] |= (filled_state.update(arith, new_state) == ERROR_SUCCESS) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.no_accounts == 3U) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].address == new_account.address) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].balance == new_account.balance) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].nonce == new_account.nonce) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (filled_state.accounts[2].byte_code.size == new_account.byte_code.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].byte_code[0] == new_account.byte_code[0]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].byte_code[1] == new_account.byte_code[1]) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].storage.size == new_account.storage.size) ? ERROR_SUCCESS : __LINE__;
    result[instance] |=
        (filled_state.accounts[2].storage.capacity == new_account.storage.capacity) ? ERROR_SUCCESS : __LINE__;
    result[instance] |= (filled_state.accounts[2].storage.storage[0].key == new_account.storage.storage[0].key)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (filled_state.accounts[2].storage.storage[0].value == new_account.storage.storage[0].value)
                            ? ERROR_SUCCESS
                            : __LINE__;
    result[instance] |= (filled_state.flags[2].flags == CuEVM::ACCOUNT_ALL_FLAG) ? ERROR_SUCCESS : __LINE__;

    // send back the fill state
    gpu_states[instance] = filled_state;
}

#ifdef GPU
TEST_F(StateAccessTest, StateAccessOperationsGPU) {
    CuEVM::state_access_t* cpu_states = CuEVM::state_access_t::get_cpu(2);

    CUDA_CHECK(hipDeviceReset());
    CuEVM::state_access_t* gpu_states = CuEVM::state_access_t::get_gpu_from_cpu(cpu_states, 2);
    uint32_t* d_result;
    hipMalloc((void**)&d_result, 2 * sizeof(uint32_t));
    test_state_access_kernel<<<2, CuEVM::cgbn_tpi>>>(gpu_states, d_result, 2);
    CUDA_CHECK(hipDeviceSynchronize());

    uint32_t* h_result = (uint32_t*)malloc(2 * sizeof(uint32_t));
    CUDA_CHECK(hipMemcpy(h_result, d_result, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    for (int i = 0; i < 2; i++) {
        EXPECT_EQ(h_result[i], ERROR_SUCCESS);
    }
    free(h_result);
    CUDA_CHECK(hipFree(d_result));
    CuEVM::state_access_t* filled_states = CuEVM::state_access_t::get_cpu_from_gpu(gpu_states, 2);
    for (int i = 0; i < 2; i++) {
        EXPECT_EQ(filled_states[i].no_accounts, 3U);
        EXPECT_EQ(filled_states[i].accounts[0].address, 0x12345678U);
        EXPECT_EQ(filled_states[i].accounts[0].balance, 0x100U);
        EXPECT_EQ(filled_states[i].accounts[0].nonce, 0x2U);
        EXPECT_EQ(filled_states[i].accounts[0].byte_code.size, 2U);
        EXPECT_EQ(filled_states[i].accounts[0].byte_code[0], 0x11);
        EXPECT_EQ(filled_states[i].accounts[0].byte_code[1], 0x22);
        EXPECT_EQ(filled_states[i].accounts[0].storage.size, 2U);
        EXPECT_EQ(filled_states[i].accounts[0].storage.capacity, 4U);
        EXPECT_EQ(filled_states[i].accounts[0].storage.storage[0].key, 0x1U);
        EXPECT_EQ(filled_states[i].accounts[0].storage.storage[0].value, 0x3U);
        EXPECT_EQ(filled_states[i].accounts[0].storage.storage[1].key, 0x2U);
        EXPECT_EQ(filled_states[i].accounts[0].storage.storage[1].value, 0x4U);
        EXPECT_EQ(filled_states[i].accounts[1].address, 0x87654321U);
        EXPECT_EQ(filled_states[i].accounts[1].balance, 0U);
        EXPECT_EQ(filled_states[i].accounts[1].nonce, 0U);
        EXPECT_EQ(filled_states[i].accounts[1].byte_code.size, 0U);
        EXPECT_EQ(filled_states[i].accounts[1].byte_code.data, nullptr);
        EXPECT_EQ(filled_states[i].accounts[1].storage.size, 0U);
        EXPECT_EQ(filled_states[i].accounts[1].storage.capacity, 0U);
        EXPECT_EQ(filled_states[i].accounts[1].storage.storage, nullptr);
        EXPECT_EQ(filled_states[i].accounts[2].address, 0x87651234U);
        EXPECT_EQ(filled_states[i].accounts[2].balance, 0U);
        EXPECT_EQ(filled_states[i].accounts[2].nonce, 0U);
        EXPECT_EQ(filled_states[i].accounts[2].byte_code.size, 3U);
        EXPECT_EQ(filled_states[i].accounts[2].byte_code[0], 0xFFU);
        EXPECT_EQ(filled_states[i].accounts[2].byte_code[1], 0x00U);
        EXPECT_EQ(filled_states[i].accounts[2].byte_code[2], 0x11U);
        EXPECT_EQ(filled_states[i].accounts[2].storage.size, 1U);
        EXPECT_EQ(filled_states[i].accounts[2].storage.capacity, 4U);
        EXPECT_EQ(filled_states[i].accounts[2].storage.storage[0].key, 0x1U);
        EXPECT_EQ(filled_states[i].accounts[2].storage.storage[0].value, 0x3U);
        EXPECT_EQ(filled_states[i].flags[2].flags, CuEVM::ACCOUNT_ALL_FLAG);
    }
    CuEVM::state_access_t::cpu_free(cpu_states, 2);
    // CuEVM::state_access_t::cpu_free(filled_states, 2);
    CUDA_CHECK(hipDeviceReset());
}
#endif