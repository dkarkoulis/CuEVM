#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <stdint.h>
#include <stdlib.h>

#include <CuEVM/core/byte_array.cuh>
#include <CuEVM/utils/evm_defines.cuh>
#include <CuEVM/utils/error_codes.cuh>

TEST(ByteArrayTests, ConstructorWithHexString) {
    CuEVM::byte_array_t byteArray("0A1B2C");
    ASSERT_EQ(byteArray.size, 3);
    ASSERT_EQ(byteArray[0], 0x0A);
    ASSERT_EQ(byteArray[1], 0x1B);
    ASSERT_EQ(byteArray[2], 0x2C);
}

TEST(ByteArrayTests, ConstructorWithHexStringAndFixedSize) {
    CuEVM::byte_array_t byteArray("0A1B2C", 5U);
    ASSERT_EQ(byteArray.size, 5);
    ASSERT_EQ(byteArray[0], 0x0A);
    ASSERT_EQ(byteArray[1], 0x1B);
    ASSERT_EQ(byteArray[2], 0x2C);
    ASSERT_EQ(byteArray[3], 0x00);  // Assuming zero padding
    ASSERT_EQ(byteArray[4], 0x00);  // Assuming zero padding
}

TEST(ByteArrayTests, CopyConstructor) {
    CuEVM::byte_array_t byteArray1("0A1B2C");
    CuEVM::byte_array_t byteArray2(byteArray1);
    ASSERT_EQ(byteArray2.size, 3);
    ASSERT_EQ(byteArray2[0], 0x0A);
    ASSERT_EQ(byteArray2[1], 0x1B);
    ASSERT_EQ(byteArray2[2], 0x2C);
}

TEST(ByteArrayTests, AssignmentOperator) {
    CuEVM::byte_array_t byteArray1("0A1B2C");
    CuEVM::byte_array_t byteArray2 = byteArray1;
    ASSERT_EQ(byteArray2.size, 3);
    ASSERT_EQ(byteArray2[0], 0x0A);
    ASSERT_EQ(byteArray2[1], 0x1B);
    ASSERT_EQ(byteArray2[2], 0x2C);
}

TEST(ByteArrayTests, GrowFunction) {
    CuEVM::byte_array_t byteArray("0A1B2C");
    byteArray.grow(5, 1);  // Grow with zero padding
    ASSERT_EQ(byteArray.size, 5);
    ASSERT_EQ(byteArray[0], 0x0A);
    ASSERT_EQ(byteArray[1], 0x1B);
    ASSERT_EQ(byteArray[2], 0x2C);
    ASSERT_EQ(byteArray[3], 0x00);  // Zero padding
    ASSERT_EQ(byteArray[4], 0x00);  // Zero padding
}

TEST(ByteArrayTests, PrintFunction) {
    CuEVM::byte_array_t byteArray("0A1B2C");
    testing::internal::CaptureStdout();
    byteArray.print();
    std::string output = testing::internal::GetCapturedStdout();
    ASSERT_EQ(output,
              "size: 3\ndata: 0a1b2c\n");  // Assuming print outputs the hex
                                           // string followed by a newline
}

TEST(ByteArrayTests, DefaultConstructor) {
    CuEVM::byte_array_t byteArray;
    EXPECT_EQ(byteArray.size, 0);
    EXPECT_EQ(byteArray.data, nullptr);
}

TEST(ByteArrayTests, ConstructorWithSize) {
    CuEVM::byte_array_t byteArray(10);
    EXPECT_EQ(byteArray.size, 10);
    EXPECT_NE(byteArray.data, nullptr);
}

TEST(ByteArrayTests, ConstructorWithData) {
    uint8_t data[5] = {1, 2, 3, 4, 5};
    CuEVM::byte_array_t byteArray(data, 5);
    EXPECT_EQ(byteArray.size, 5);
    EXPECT_NE(byteArray.data, nullptr);
    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(byteArray.data[i], data[i]);
    }
}

TEST(ByteArrayTests, CopyConstructor2) {
    uint8_t data[5] = {1, 2, 3, 4, 5};
    CuEVM::byte_array_t byteArray1(data, 5);
    CuEVM::byte_array_t byteArray2(byteArray1);
    EXPECT_EQ(byteArray2.size, 5);
    EXPECT_NE(byteArray2.data, nullptr);
    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(byteArray2.data[i], data[i]);
    }
}

TEST(ByteArrayTests, AssignmentOperator2) {
    uint8_t data1[5] = {1, 2, 3, 4, 5};
    CuEVM::byte_array_t byteArray1(data1, 5);
    CuEVM::byte_array_t byteArray2;
    byteArray2 = byteArray1;
    EXPECT_EQ(byteArray2.size, 5);
    EXPECT_NE(byteArray2.data, nullptr);
    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(byteArray2.data[i], data1[i]);
    }
}

TEST(ByteArrayTests, Grow) {
    CuEVM::byte_array_t byteArray(5);
    byteArray.grow(10, 1);
    EXPECT_EQ(byteArray.size, 10);
    for (int i = 5; i < 10; ++i) {
        EXPECT_EQ(byteArray.data[i], 0);
    }
}

TEST(ByteArrayTests, ToHex) {
    uint8_t data[3] = {0x12, 0x34, 0x56};
    CuEVM::byte_array_t byteArray(data, 3);
    char* hexString = byteArray.to_hex();
    EXPECT_STREQ(hexString, "0x123456");
    free(hexString);
}

TEST(ByteArrayTests, FromHexSetLE) {
    CuEVM::byte_array_t byteArray(3);
    byteArray.from_hex("3456", LITTLE_ENDIAN, CuEVM::NO_PADDING);
    EXPECT_EQ(byteArray.size, 3);
    EXPECT_EQ(byteArray.data[0], 0x34);
    EXPECT_EQ(byteArray.data[1], 0x56);
    EXPECT_EQ(byteArray.data[2], 0x00);
}

TEST(ByteArrayTests, FromHexSetBE_LP) {
    CuEVM::byte_array_t byteArray(3);
    byteArray.from_hex("3456", BIG_ENDIAN, CuEVM::LEFT_PADDING);
    EXPECT_EQ(byteArray.size, 3);
    EXPECT_EQ(byteArray.data[0], 0x56);
    EXPECT_EQ(byteArray.data[1], 0x34);
    EXPECT_EQ(byteArray.data[2], 0x00);
}
TEST(ByteArrayTests, FromHexSetBE_RP) {
    CuEVM::byte_array_t byteArray(3);
    byteArray.from_hex("3456", BIG_ENDIAN, CuEVM::RIGHT_PADDING);
    EXPECT_EQ(byteArray.size, 3);
    EXPECT_EQ(byteArray.data[0], 0x00);
    EXPECT_EQ(byteArray.data[1], 0x56);
    EXPECT_EQ(byteArray.data[2], 0x34);
}

// Additional tests

TEST(ByteArrayTests, ConstructorWithHexString2) {
    CuEVM::byte_array_t byteArray("123456", LITTLE_ENDIAN, CuEVM::NO_PADDING);
    EXPECT_EQ(byteArray.size, 3);
    EXPECT_EQ(byteArray.data[0], 0x12);
    EXPECT_EQ(byteArray.data[1], 0x34);
    EXPECT_EQ(byteArray.data[2], 0x56);
}

TEST(ByteArrayTests, ConstructorWithHexStringAndSize) {
    CuEVM::byte_array_t byteArray("123456", 4, LITTLE_ENDIAN,
                                  CuEVM::LEFT_PADDING);
    EXPECT_EQ(byteArray.size, 4);
    EXPECT_EQ(byteArray.data[0], 0x12);
    EXPECT_EQ(byteArray.data[1], 0x34);
    EXPECT_EQ(byteArray.data[2], 0x56);
    EXPECT_EQ(byteArray.data[3], 0x00);
}

TEST(ByteArrayTests, PaddedCopyBE) {
    uint8_t data[3] = {0x12, 0x34, 0x56};
    CuEVM::byte_array_t src(data, 3);
    CuEVM::byte_array_t dst(5);
    dst.padded_copy_BE(src);
    EXPECT_EQ(dst.size, 5);
    EXPECT_EQ(dst.data[0], 0x12);
    EXPECT_EQ(dst.data[1], 0x34);
    EXPECT_EQ(dst.data[2], 0x56);
    EXPECT_EQ(dst.data[3], 0x00);
    EXPECT_EQ(dst.data[4], 0x00);
}

TEST(ByteArrayTests, FromHex) {
    CuEVM::byte_array_t byteArray;
    byteArray.from_hex("123456", LITTLE_ENDIAN, CuEVM::NO_PADDING, 0);
    EXPECT_EQ(byteArray.size, 3);
    EXPECT_EQ(byteArray.data[0], 0x12);
    EXPECT_EQ(byteArray.data[1], 0x34);
    EXPECT_EQ(byteArray.data[2], 0x56);
}

TEST(ByteArrayTests, IndexOperator) {
    uint8_t data[3] = {0x12, 0x34, 0x56};
    CuEVM::byte_array_t byteArray(data, 3);
    EXPECT_EQ(byteArray[0], 0x12);
    EXPECT_EQ(byteArray[1], 0x34);
    EXPECT_EQ(byteArray[2], 0x56);
}

TEST(ByteArrayTests, ArrayOfZeroByteArrays) {
    CuEVM::byte_array_t byteArray[5];
    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(byteArray[i].size, 0);
        EXPECT_EQ(byteArray[i].data, nullptr);
    }
}

TEST(ByteArrayTests, HasValueFunction) {
    CuEVM::byte_array_t byteArray("0A1B2C");
    EXPECT_EQ(byteArray.has_value(0x0A), ERROR_SUCCESS);
    EXPECT_EQ(byteArray.has_value(0x1B), ERROR_SUCCESS);
    EXPECT_EQ(byteArray.has_value(0x2C), ERROR_SUCCESS);
    EXPECT_EQ(byteArray.has_value(0x00), ERROR_VALUE_NOT_FOUND);
    EXPECT_EQ(byteArray.has_value(0xFF), ERROR_VALUE_NOT_FOUND);
}

TEST(ByteArrayTests, GetCpu) {
    CuEVM::byte_array_t* cpuArray = CuEVM::byte_array_t::get_cpu(5);
    for (int i = 0; i < 5; ++i) {
        EXPECT_EQ(cpuArray[i].size, 0);
        EXPECT_EQ(cpuArray[i].data, nullptr);
    }
    CuEVM::byte_array_t::cpu_free(cpuArray, 5);
}

TEST(ByteArrayTests, CpuGpuFree) {
    CuEVM::byte_array_t* cpuArray = CuEVM::byte_array_t::get_cpu(2);
    cpuArray[0].grow(3, 1);
    cpuArray[0].data[0] = 0x12;
    cpuArray[0].data[1] = 0x34;
    cpuArray[0].data[2] = 0x56;

    cpuArray[1].grow(2, 1);
    cpuArray[1].data[0] = 0x78;
    cpuArray[1].data[1] = 0x9A;

    CUDA_CHECK(hipDeviceReset());
    CuEVM::byte_array_t* gpuArray =
        CuEVM::byte_array_t::gpu_from_cpu(cpuArray, 2);

    CuEVM::byte_array_t::gpu_free(gpuArray, 2);

    CuEVM::byte_array_t::cpu_free(cpuArray, 2);
    CUDA_CHECK(hipDeviceReset());
}

// Additional GPU tests

__global__ void testKernel(CuEVM::byte_array_t* gpuArray, uint32_t count, uint32_t *result) {
    int32_t instance =
        (blockIdx.x * blockDim.x + threadIdx.x) / CuEVM::cgbn_tpi;
    if (instance >= count) return;
    result[instance] = ERROR_SUCCESS;
    if (instance == 0) {
        gpuArray[0].grow(3, 1);
        gpuArray[0].data[0] = 0x12;
        gpuArray[0].data[1] = 0x34;
        gpuArray[0].data[2] = 0x56;
        result[instance] |= (gpuArray[0].has_value(0x12) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[0].has_value(0x34) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[0].has_value(0x56) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[0].has_value(0x00) == ERROR_VALUE_NOT_FOUND ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
    } else if (instance == 1) {
        gpuArray[1].grow(3, 1);
        gpuArray[1].data[0] = 0x78;
        gpuArray[1].data[1] = 0x9A;
        gpuArray[1].data[2] = 0xBC;
        result[instance] |= (gpuArray[1].has_value(0x78) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[1].has_value(0x9A) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[1].has_value(0xBC) == ERROR_SUCCESS ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
        result[instance] |= (gpuArray[1].has_value(0x00) == ERROR_VALUE_NOT_FOUND ? ERROR_SUCCESS : ERROR_VALUE_NOT_FOUND);
    }
}

TEST(ByteArrayTests, GpuKernelTest) {
    CuEVM::byte_array_t* cpuArray = CuEVM::byte_array_t::get_cpu(2);
    CUDA_CHECK(hipDeviceReset());
    CuEVM::byte_array_t* gpuArray =
        CuEVM::byte_array_t::gpu_from_cpu(cpuArray, 2);
    uint32_t *d_result;
    hipMalloc(&d_result, 2 * sizeof(uint32_t));
    testKernel<<<2, CuEVM::cgbn_tpi>>>(gpuArray, 2, d_result);
    CUDA_CHECK(hipDeviceSynchronize());
    CuEVM::byte_array_t* results =
        CuEVM::byte_array_t::cpu_from_gpu(gpuArray, 2);
    CuEVM::byte_array_t* expectedCpuArray = CuEVM::byte_array_t::get_cpu(2);
    expectedCpuArray[0].grow(3, 1);
    expectedCpuArray[0].data[0] = 0x12;
    expectedCpuArray[0].data[1] = 0x34;
    expectedCpuArray[0].data[2] = 0x56;

    expectedCpuArray[1].grow(3, 1);
    expectedCpuArray[1].data[0] = 0x78;
    expectedCpuArray[1].data[1] = 0x9A;
    expectedCpuArray[1].data[2] = 0xBC;

    for (int i = 0; i < 2; ++i) {
        EXPECT_EQ(results[i].size, expectedCpuArray[i].size);
        for (int j = 0; j < results[i].size; ++j) {
            EXPECT_EQ(results[i].data[j], expectedCpuArray[i].data[j]);
        }
    }
    uint32_t *h_result;
    h_result = (uint32_t *)malloc(2 * sizeof(uint32_t));
    CUDA_CHECK(hipMemcpy(h_result, d_result, 2 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    for (int i = 0; i < 2; i++) {
        EXPECT_EQ(h_result[i], ERROR_SUCCESS);
    }
    free(h_result);
    CUDA_CHECK(hipFree(d_result));
    CuEVM::byte_array_t::cpu_free(cpuArray, 2);
    CuEVM::byte_array_t::cpu_free(expectedCpuArray, 2);
    CuEVM::byte_array_t::cpu_free(results, 2);
    CUDA_CHECK(hipDeviceReset());
}