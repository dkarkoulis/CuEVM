#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

#include <CuEVM/utils/arith.cuh>
#include <CuEVM/utils/error_codes.cuh>
#include <CuEVM/utils/evm_utils.cuh>

using namespace CuEVM;

__global__ void kernel_test_cgbn_set_memory(uint32_t *result) {
    ArithEnv arithEnv(cgbn_no_checks);

    __SHARED_MEMORY__ uint8_t *src;
    __ONE_GPU_THREAD_BEGIN__
    src = new uint8_t[32]{0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    __ONE_GPU_THREAD_END__
    bn_t dst;
    cgbn_set_memory(arithEnv.env, dst, src, 32);
    *result = cgbn_get_ui32(arithEnv.env, dst);
    __ONE_GPU_THREAD_BEGIN__
    delete src;
    __ONE_GPU_THREAD_END__
}

TEST(ArithTests, TestCgbnSetMemoryCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    uint8_t src[32] = {0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    bn_t dst;
    cgbn_set_memory(arithEnv.env, dst, src, 32);

    uint32_t result;
    result = cgbn_get_ui32(arithEnv.env, dst);
    EXPECT_EQ(result, 0x9ABCDEF0);
}

#ifdef GPU
TEST(ArithTests, TestCgbnSetMemory) {
    uint32_t *d_result;
    uint32_t h_result;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint32_t));

    kernel_test_cgbn_set_memory<<<1, CuEVM::cgbn_tpi>>>(d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x9ABCDEF0);
    hipFree(d_result);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestCgbnSetSizeTCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    size_t src = 0x123456789ABCDEF0;
    bn_t dst;
    cgbn_set_size_t(arithEnv.env, dst, src);
    uint64_t result;
    cgbn_get_uint64_t(arithEnv.env, result, dst);
    EXPECT_EQ(result, 0x123456789ABCDEF0);
}

__global__ void kernel_test_cgbn_set_size_t(uint64_t *result) {
    ArithEnv arithEnv(cgbn_no_checks);

    size_t src = 0x123456789ABCDEF0;
    bn_t dst;
    cgbn_set_size_t(arithEnv.env, dst, src);
    cgbn_get_uint64_t(arithEnv.env, *result, dst);
}

#ifdef GPU
TEST(ArithTests, TestCgbnSetSizeT) {
    uint64_t *d_result;
    uint64_t h_result;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint64_t));

    kernel_test_cgbn_set_size_t<<<1, CuEVM::cgbn_tpi>>>(d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x123456789ABCDEF0);
    hipFree(d_result);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestCgbnGetSizeTCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    bn_t src;
    size_t expected = 0x123456789ABCDEF0;
    cgbn_set_size_t(arithEnv.env, src, expected);
    size_t result;
    EXPECT_EQ(cgbn_get_size_t(arithEnv.env, result, src), ERROR_SUCCESS);
    EXPECT_EQ(result, 0x123456789ABCDEF0);
}

__global__ void kernel_test_cgbn_get_size_t(size_t *result,
                                            int32_t *error_code) {
    ArithEnv arithEnv(cgbn_no_checks);

    bn_t src;
    size_t expected = 0x123456789ABCDEF0;
    cgbn_set_size_t(arithEnv.env, src, expected);

    *error_code = cgbn_get_size_t(arithEnv.env, *result, src);
}

#ifdef GPU
TEST(ArithTests, TestCgbnGetSizeT) {
    size_t *d_result;
    int32_t *d_error_code;
    size_t h_result;
    int32_t h_error_code;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(size_t));
    hipMalloc(&d_error_code, sizeof(int32_t));

    kernel_test_cgbn_get_size_t<<<1, CuEVM::cgbn_tpi>>>(d_result, d_error_code);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(size_t), hipMemcpyDeviceToHost);
    hipMemcpy(&h_error_code, d_error_code, sizeof(int32_t),
               hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x123456789ABCDEF0);
    EXPECT_EQ(h_error_code, ERROR_SUCCESS);
    hipFree(d_result);
    hipFree(d_error_code);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestCgbnGetUint64TCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    bn_t src;
    uint64_t expected = 0x123456789ABCDEF0;
    cgbn_set_size_t(arithEnv.env, src, expected);
    uint64_t result;
    EXPECT_EQ(cgbn_get_uint64_t(arithEnv.env, result, src), ERROR_SUCCESS);
    EXPECT_EQ(result, 0x123456789ABCDEF0);
}

__global__ void kernel_test_cgbn_get_uint64_t(uint64_t *result,
                                              int32_t *error_code) {
    ArithEnv arithEnv(cgbn_no_checks);

    bn_t src;
    uint64_t expected = 0x123456789ABCDEF0;
    cgbn_set_size_t(arithEnv.env, src, expected);

    *error_code = cgbn_get_uint64_t(arithEnv.env, *result, src);
}

#ifdef GPU
TEST(ArithTests, TestCgbnGetUint64T) {
    uint64_t *d_result;
    int32_t *d_error_code;
    uint64_t h_result;
    int32_t h_error_code;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint64_t));
    hipMalloc(&d_error_code, sizeof(int32_t));

    kernel_test_cgbn_get_uint64_t<<<1, CuEVM::cgbn_tpi>>>(d_result,
                                                          d_error_code);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(&h_error_code, d_error_code, sizeof(int32_t),
               hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x123456789ABCDEF0);
    EXPECT_EQ(h_error_code, ERROR_SUCCESS);
    hipFree(d_result);
    hipFree(d_error_code);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestCgbnGetUint32TCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    bn_t src;
    uint32_t expected = 0x12345678;
    cgbn_set_ui32(arithEnv.env, src, expected);

    uint32_t result;
    EXPECT_EQ(cgbn_get_uint32_t(arithEnv.env, result, src), ERROR_SUCCESS);
    EXPECT_EQ(result, 0x12345678);
}

__global__ void kernel_test_cgbn_get_uint32_t(uint32_t *result,
                                              int32_t *error_code) {
    ArithEnv arithEnv(cgbn_no_checks);

    bn_t src;
    uint32_t expected = 0x12345678;
    cgbn_set_ui32(arithEnv.env, src, expected);

    *error_code = cgbn_get_uint32_t(arithEnv.env, *result, src);
}

#ifdef GPU
TEST(ArithTests, TestCgbnGetUint32T) {
    uint32_t *d_result;
    int32_t *d_error_code;
    uint32_t h_result;
    int32_t h_error_code;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint32_t));
    hipMalloc(&d_error_code, sizeof(int32_t));

    kernel_test_cgbn_get_uint32_t<<<1, CuEVM::cgbn_tpi>>>(d_result,
                                                          d_error_code);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(&h_error_code, d_error_code, sizeof(int32_t),
               hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x12345678);
    EXPECT_EQ(h_error_code, ERROR_SUCCESS);
    hipFree(d_result);
    hipFree(d_error_code);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestCgbnSetByteArrayTCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);
    uint8_t data[32] = {0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                        0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                        0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                        0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};

    byte_array_t byte_array(data, 32);
    bn_t out;

    EXPECT_EQ(cgbn_set_byte_array_t(arithEnv.env, out, byte_array),
              ERROR_SUCCESS);
    EXPECT_EQ(cgbn_get_ui32(arithEnv.env, out), 0x9ABCDEF0);
}

__global__ void kernel_test_cgbn_set_byte_array_t(uint32_t *result,
                                                  int32_t *error_code) {
    ArithEnv arithEnv(cgbn_no_checks);
    __SHARED_MEMORY__ uint8_t *src;
    __ONE_GPU_THREAD_BEGIN__
    src = new uint8_t[32]{0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    __ONE_GPU_THREAD_END__

    byte_array_t byte_array(src, 32);
    bn_t out;

    *error_code = cgbn_set_byte_array_t(arithEnv.env, out, byte_array);
    *result = cgbn_get_ui32(arithEnv.env, out);

    __ONE_GPU_THREAD_BEGIN__
    delete src;
    __ONE_GPU_THREAD_END__
}

#ifdef GPU
TEST(ArithTests, TestCgbnSetByteArrayT) {
    uint32_t *d_result;
    int32_t *d_error_code;
    uint32_t h_result;
    int32_t h_error_code;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint32_t));
    hipMalloc(&d_error_code, sizeof(int32_t));

    kernel_test_cgbn_set_byte_array_t<<<1, CuEVM::cgbn_tpi>>>(d_result,
                                                              d_error_code);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipMemcpy(&h_error_code, d_error_code, sizeof(int32_t),
               hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, 0x9ABCDEF0);
    EXPECT_EQ(h_error_code, ERROR_SUCCESS);
    hipFree(d_result);
    hipFree(d_error_code);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestGetSubByteArrayTCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);
    uint8_t data[8] = {0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};

    byte_array_t byte_array(data, 8);
    bn_t index, length;
    cgbn_set_ui32(arithEnv.env, index, 2);
    cgbn_set_ui32(arithEnv.env, length, 4);
    byte_array_t out;

    EXPECT_EQ(get_sub_byte_array_t(arithEnv, byte_array, index, length, out),
              ERROR_SUCCESS);
    for (uint32_t i = 0; i < 4; ++i) {
        EXPECT_EQ(out.data[i], data[i + 2]);
    }
}

__global__ void kernel_test_get_sub_byte_array_t(int32_t *error_code,
                                                 uint8_t *result) {
    ArithEnv arithEnv(cgbn_no_checks);

    __SHARED_MEMORY__ uint8_t *data;
    __ONE_GPU_THREAD_BEGIN__
    data = new uint8_t[8]{0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    __ONE_GPU_THREAD_END__

    byte_array_t byte_array(data, 8);
    bn_t index, length;
    cgbn_set_ui32(arithEnv.env, index, 2);
    cgbn_set_ui32(arithEnv.env, length, 4);
    byte_array_t out;

    *error_code =
        get_sub_byte_array_t(arithEnv, byte_array, index, length, out);

    for (int i = 0; i < 4; ++i) {
        result[i] = out.data[i];
    }

    __ONE_GPU_THREAD_BEGIN__
    delete data;
    __ONE_GPU_THREAD_END__
}

#ifdef GPU
TEST(ArithTests, TestGetSubByteArrayT) {
    int32_t *d_error_code;
    uint8_t *d_result;
    int32_t h_error_code;
    uint8_t h_result[4];
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_error_code, sizeof(int32_t));
    hipMalloc(&d_result, 4 * sizeof(uint8_t));

    kernel_test_get_sub_byte_array_t<<<1, CuEVM::cgbn_tpi>>>(d_error_code,
                                                             d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_error_code, d_error_code, sizeof(int32_t),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_result, d_result, 4 * sizeof(uint8_t), hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result[0], 0x56);
    EXPECT_EQ(h_result[1], 0x78);
    EXPECT_EQ(h_result[2], 0x9A);
    EXPECT_EQ(h_result[3], 0xBC);
    EXPECT_EQ(h_error_code, ERROR_SUCCESS);
    hipFree(d_error_code);
    hipFree(d_result);
    CUDA_CHECK(hipDeviceReset());
}
#endif

TEST(ArithTests, TestEvmAddressConversionCPU) {
    ArithEnv arithEnv(cgbn_no_checks, 0);

    uint8_t src[32] = {0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                       0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    bn_t address;
    cgbn_set_memory(arithEnv.env, address, src, 32);

    evm_address_conversion(arithEnv, address);

    evm_word_t result;
    cgbn_store(arithEnv.env, (cgbn_evm_word_t_ptr)&result, address);

    // address are 20 bytes long, so the first 12 bytes should be 0x00 12 = 3 *
    // 4 bytes
    EXPECT_EQ(result._limbs[7], 0x00);
    EXPECT_EQ(result._limbs[6], 0x00);
    EXPECT_EQ(result._limbs[5], 0x00);
    EXPECT_EQ(result._limbs[4], 0x9ABCDEF0);
    EXPECT_EQ(result._limbs[3], 0x12345678);
    EXPECT_EQ(result._limbs[2], 0x9ABCDEF0);
    EXPECT_EQ(result._limbs[1], 0x12345678);
    EXPECT_EQ(result._limbs[0], 0x9ABCDEF0);
}

__global__ void kernel_test_evm_address_conversion(uint32_t *result) {
    ArithEnv arithEnv(cgbn_no_checks);

    __SHARED_MEMORY__ uint8_t *src;
    __ONE_GPU_THREAD_BEGIN__
    src = new uint8_t[32]{0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0,
                          0x12, 0x34, 0x56, 0x78, 0x9A, 0xBC, 0xDE, 0xF0};
    __ONE_GPU_THREAD_END__
    bn_t address;
    cgbn_set_memory(arithEnv.env, address, src, 32);
    evm_address_conversion(arithEnv, address);

    __SHARED_MEMORY__ evm_word_t result_evm_word;
    cgbn_store(arithEnv.env, (cgbn_evm_word_t_ptr)&result_evm_word, address);
    *result |= result_evm_word._limbs[7] == 0x00 ? ERROR_SUCCESS : __LINE__;
    *result |= result_evm_word._limbs[6] == 0x00 ? ERROR_SUCCESS : __LINE__;
    *result |= result_evm_word._limbs[5] == 0x00 ? ERROR_SUCCESS : __LINE__;
    *result |=
        result_evm_word._limbs[4] == 0x9ABCDEF0 ? ERROR_SUCCESS : __LINE__;
    *result |=
        result_evm_word._limbs[3] == 0x12345678 ? ERROR_SUCCESS : __LINE__;
    *result |=
        result_evm_word._limbs[2] == 0x9ABCDEF0 ? ERROR_SUCCESS : __LINE__;
    *result |=
        result_evm_word._limbs[1] == 0x12345678 ? ERROR_SUCCESS : __LINE__;
    *result |=
        result_evm_word._limbs[0] == 0x9ABCDEF0 ? ERROR_SUCCESS : __LINE__;

    __ONE_GPU_THREAD_BEGIN__
    delete src;
    __ONE_GPU_THREAD_END__
}

#ifdef GPU
TEST(ArithTests, TestEvmAddressConversion) {
    uint32_t *d_result;
    uint32_t h_result;
    CUDA_CHECK(hipDeviceReset());
    hipMalloc(&d_result, sizeof(uint32_t));

    kernel_test_evm_address_conversion<<<1, CuEVM::cgbn_tpi>>>(d_result);
    hipDeviceSynchronize();
    hipMemcpy(&h_result, d_result, sizeof(uint32_t), hipMemcpyDeviceToHost);

    EXPECT_EQ(h_result, ERROR_SUCCESS);
    hipFree(d_result);
    CUDA_CHECK(hipDeviceReset());
}
#endif