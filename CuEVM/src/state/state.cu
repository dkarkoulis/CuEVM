// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation Programme)
// Author: Stefan-Dan Ciocirlan
// Data: 2023-11-30
// SPDX-License-Identifier: MIT

#include <CuEVM/state/state.cuh>
#include <CuEVM/utils/error_codes.cuh>
#include <CuCrypto/keccak.cuh>

namespace CuEVM {
    namespace state {
        __host__ __device__ state_t::~state_t() {
            free();
        }


        __host__ __device__ state_t::state_t(const state_t &other) {
            duplicate(other);
        }


        __host__ state_t::state_t(const cJSON *json, int32_t managed ) {
            no_accounts = 0;
            accounts = nullptr;
            from_json(json, managed);
        }


        __host__ __device__ state_t& state_t::operator=(const state_t &other) {
            if (this != &other) {
                free();
                duplicate(other);
            }
            return *this;
        }

        __host__ __device__ void state_t::duplicate(const state_t &other) {
            no_accounts = other.no_accounts;
            if (no_accounts > 0) {
                accounts = new CuEVM::account::account_t[no_accounts];
                for (uint32_t idx = 0; idx < no_accounts; idx++) {
                    accounts[idx] = other.accounts[idx];
                }
            } else {
                accounts = nullptr;
            }
        }


        __host__ __device__ void state_t::free() {
            if (accounts != nullptr && no_accounts > 0) {
                delete[] accounts;
            }
            no_accounts = 0;
            accounts = nullptr;
        }
        __host__ __device__ int32_t state_t::get_account_index(
                ArithEnv &arith,
                const bn_t &address,
                uint32_t &index) {
            for (index = 0; index < no_accounts; index++) {
                if (accounts[index].has_address(arith, address)) {
                    return ERROR_SUCCESS;
                }
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }


        __host__ __device__ int32_t state_t::get_account(
            ArithEnv &arith,
            const bn_t &address,
            CuEVM::account::account_t &account
        ) {
            uint32_t index;
            if (get_account_index(arith, address, index) == ERROR_SUCCESS) {
                account = accounts[index];
                return ERROR_SUCCESS;
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }

        __host__ __device__ int32_t state_t::get_account(
            ArithEnv &arith,
            const bn_t &address,
            CuEVM::account::account_t* &account_ptr) {
            uint32_t index;
            if (get_account_index(arith, address, index) == ERROR_SUCCESS) {
                account_ptr = &accounts[index];
                return ERROR_SUCCESS;
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;

        }


        __host__ __device__ int32_t state_t::add_account(
            const CuEVM::account::account_t &account
        ) {
            CuEVM::account::account_t *tmp_accounts = new CuEVM::account::account_t[no_accounts + 1];
            std::copy(accounts, accounts + no_accounts, tmp_accounts);
            tmp_accounts[no_accounts] = account;
            if (accounts != nullptr) {
                delete[] accounts;
            }
            accounts = tmp_accounts;
            no_accounts++;
            return ERROR_SUCCESS;
        }


        __host__ __device__ int32_t state_t::set_account(
            ArithEnv &arith,
            const CuEVM::account::account_t &account
        ) {
            bn_t target_address;
            cgbn_load(arith.env, target_address, (cgbn_evm_word_t_ptr) &(account.address));
            for (uint32_t idx = 0; idx < no_accounts; idx++) {
                if (accounts[idx].has_address(arith, target_address)) {
                    accounts[idx] = account;
                    return ERROR_SUCCESS;
                }
            }

            return add_account(account);
        }


        __host__ __device__ int32_t state_t::has_account(
            ArithEnv &arith,
            const bn_t &address
        ) {
            for (uint32_t idx = 0; idx < no_accounts; idx++) {
                if (accounts[idx].has_address(arith, address)) {
                    return 1;
                }
            }
            return 0;
        }


        __host__ __device__ int32_t state_t::update_account(
            ArithEnv &arith,
            const CuEVM::account::account_t &account
        ) {
            bn_t target_address;
            cgbn_load(arith.env, target_address, (cgbn_evm_word_t_ptr) &(account.address));
            for (uint32_t idx = 0; idx < no_accounts; idx++) {
                if (accounts[idx].has_address(arith, target_address)) {
                    accounts[idx].update(arith, account);
                    return ERROR_SUCCESS;
                }
            }
            return add_account(account);
        }


        __host__ int32_t state_t::from_json(const cJSON *state_json, int32_t managed) {
            free();
            //if (!cJSON_IsArray(state_json)) return 0;
            no_accounts = cJSON_GetArraySize(state_json);
            if (no_accounts == 0) return 1;
            if (managed) {
                CUDA_CHECK(hipMallocManaged(
                    (void **)&(accounts),
                    no_accounts * sizeof(CuEVM::account::account_t)
                ));
            } else {
                accounts = new CuEVM::account::account_t[no_accounts];
            }
            uint32_t idx = 0;
            cJSON *account_json;
            cJSON_ArrayForEach(account_json, state_json)
            {
                accounts[idx++].from_json(account_json, managed);
            }
            return 1;
        }


        __host__ __device__ void state_t::print() {
            printf("no_accounts: %lu\n", no_accounts);
            for (uint32_t idx = 0; idx < no_accounts; idx++) {
                printf("accounts[%lu]:\n", idx);
                accounts[idx].print();
            }
        }


        __host__ cJSON* state_t::to_json() {
            cJSON *state_json = nullptr;
            cJSON *account_json = nullptr;
            char *hex_string_ptr = new char[CuEVM::word_size * 2 + 3];
            char *flag_string_ptr = nullptr;
            state_json = cJSON_CreateObject();
            for(uint32_t idx = 0; idx < no_accounts; idx++) {
                accounts[idx].address.to_hex(hex_string_ptr, 0, 5);
                account_json = accounts[idx].to_json();
                cJSON_AddItemToObject(
                    state_json,
                    hex_string_ptr,
                    account_json);
            }
            delete[] hex_string_ptr;
            hex_string_ptr = nullptr;
            return state_json;
        }



        __host__ __device__ void state_access_t::duplicate(const state_access_t &other) {
            state_t::duplicate(other);
            if (no_accounts > 0) {
                flags = new CuEVM::account::account_flags_t[no_accounts];
                std::copy(other.flags, other.flags + no_accounts, flags);
            } else {
                flags = nullptr;
            }
        }

        __host__ __device__ void state_access_t::free() {
            if (flags != nullptr && no_accounts > 0) {
                delete[] flags;
                flags = nullptr;
            }
            state_t::free();
        }

        __host__ __device__ int32_t state_access_t::get_account(
            ArithEnv &arith,
            const bn_t &address,
            CuEVM::account::account_t &account,
            const CuEVM::account::account_flags_t flag) {
            uint32_t index = 0;
            if(state_t::get_account_index(arith, address, index)) {
                flags[index].update(flag);
                account = accounts[index];
                return ERROR_SUCCESS;
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }


        __host__ __device__ int32_t state_access_t::get_account(
            ArithEnv &arith,
            const bn_t &address,
            CuEVM::account::account_t* &account_ptr,
            const CuEVM::account::account_flags_t flag) {
            uint32_t index = 0;
            if(state_t::get_account_index(arith, address, index) == ERROR_SUCCESS) {
                flags[index].update(flag);
                account_ptr = &accounts[index];
                return ERROR_SUCCESS;
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }

        __host__ __device__ int32_t state_access_t::add_account(
            const CuEVM::account::account_t &account,
            const CuEVM::account::account_flags_t flag) {
            state_t::add_account(account);
            uint32_t index = no_accounts - 1;
            CuEVM::account::account_flags_t *tmp_flags = new CuEVM::account::account_flags_t[no_accounts];
            std::copy(flags, flags + no_accounts - 1, tmp_flags);
            if (flags != nullptr) {
                delete[] flags;
            }
            flags = tmp_flags;
            flags[index] = flag;
            return ERROR_SUCCESS;
        }

        __host__ __device__ int32_t state_access_t::add_duplicate_account(
            CuEVM::account::account_t* &account_ptr,
            CuEVM::account::account_t* &src_account_ptr,
            const CuEVM::account::account_flags_t flag) {
            CuEVM::account::account_flags_t no_storage_copy(ACCOUNT_NON_STORAGE_FLAG);
            uint32_t index = 0;
            account_ptr = new CuEVM::account::account_t(
                src_account_ptr,
                no_storage_copy);
            return add_account(*account_ptr, flag);
        }

        __host__ __device__ int32_t state_access_t::add_new_account(
            ArithEnv &arith,
            const bn_t &address,
            CuEVM::account::account_t* &account_ptr,
            const CuEVM::account::account_flags_t flag) {
            uint32_t index = 0;
            account_ptr = new CuEVM::account::account_t(
                arith,
                address);
            return add_account(*account_ptr, flag);
        }

        __host__ __device__ int32_t state_access_t::set_account(
            ArithEnv &arith,
            const CuEVM::account::account_t &account,
            const CuEVM::account::account_flags_t flag) {
            uint32_t index = 0;
            if (update_account(arith, account, flag)) {
                return add_account(account, flag);
            } else {
                return ERROR_SUCCESS;
            }
        }

        __host__ __device__ int32_t state_access_t::update_account(
            ArithEnv &arith,
            const CuEVM::account::account_t &account,
            const CuEVM::account::account_flags_t flag) {
            bn_t target_address;
            cgbn_load(arith.env, target_address, (cgbn_evm_word_t_ptr) &(account.address));
            uint32_t index = 0;
            if(state_t::get_account_index(arith, target_address, index) == ERROR_SUCCESS) {
                accounts[index].update(arith, account, flag);
                if (flags != nullptr){
                    flags[index].update(flag);
                }

                return ERROR_SUCCESS;
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }

        __host__ __device__ int32_t state_access_t::update(
            ArithEnv &arith,
            const state_access_t &other) {
            uint32_t index = 0;
            for (uint32_t i = 0; i < other.no_accounts; i++) {
                // if update failed (not exist), add the account
                if (update_account(arith, other.accounts[i], other.flags[i]) != ERROR_SUCCESS) {
                    add_account(other.accounts[i], index);
                    flags[no_accounts - 1] = other.flags[i];
                }
            }
            return ERROR_SUCCESS;
        }



        __host__ __device__ void state_access_t::print() {
            printf("no_accounts: %lu\n", no_accounts);
            for (uint32_t idx = 0; idx < no_accounts; idx++) {
                printf("accounts[%lu]:\n", idx);
                accounts[idx].print();
                printf("flags[%lu]:\n", idx);
                flags[idx].print();
            }
        }

        __host__ cJSON* state_access_t::to_json() {
            cJSON *state_json = nullptr;
            cJSON *account_json = nullptr;
            char *hex_string_ptr = new char[CuEVM::word_size * 2 + 3];
            char *flag_string_ptr = new char[sizeof(uint32_t) * 2 + 3];
            state_json = cJSON_CreateObject();
            for(uint32_t idx = 0; idx < no_accounts; idx++) {
                accounts[idx].address.to_hex(hex_string_ptr, 0, 5);
                account_json = accounts[idx].to_json();
                cJSON_AddStringToObject(
                    account_json,
                    "flags",
                    flags[idx].to_hex(flag_string_ptr)
                );
                cJSON_AddItemToObject(
                    state_json,
                    hex_string_ptr,
                    account_json);
            }
            delete[] hex_string_ptr;
            hex_string_ptr = nullptr;
            delete[] flag_string_ptr;
            flag_string_ptr = nullptr;
            return state_json;
        }

        __host__ int32_t state_access_t::get_account_index_evm(
            const evm_word_t &address,
            uint32_t &index) const {
            for (index = 0; index < no_accounts; index++) {
                if (accounts[index].address == address) {
                    return ERROR_SUCCESS;
                }
            }
            return ERROR_STATE_ADDRESS_NOT_FOUND;
        }

        __host__ cJSON* state_merge_json(
            const state_t &state1,
            const state_access_t &state2
        ) {
            cJSON *state_json = nullptr;
            cJSON *account_json = nullptr;
            cJSON *accounts_json = nullptr;


            state_json = cJSON_CreateObject();
            accounts_json = cJSON_CreateArray();
            cJSON_AddItemToObject(state_json, "accounts", accounts_json);
            uint8_t *writen_accounts;
            writen_accounts = new uint8_t[state2.no_accounts];
            std::fill(writen_accounts, writen_accounts + state2.no_accounts, 0);
            const CuEVM::account::account_t *account1_ptr = nullptr;
            const CuEVM::account::account_t *account2_ptr = nullptr;
            uint32_t jdx = 0;
            for (uint32_t idx = 0; idx < state1.no_accounts; idx++) {
                account1_ptr = &(state1.accounts[idx]);
                if (state2.get_account_index_evm(account1_ptr->address, jdx) == ERROR_SUCCESS) {
                    account2_ptr = &(state2.accounts[jdx]);
                    account_json = CuEVM::account::account_merge_json(
                        account1_ptr,
                        account2_ptr,
                        state2.flags[jdx]);
                    cJSON_AddItemToArray(accounts_json, account_json);
                    writen_accounts[jdx] = 1;
                } else {
                    account_json = CuEVM::account::account_merge_json(
                        account1_ptr,
                        account2_ptr,
                        ACCOUNT_NONE_FLAG);
                    cJSON_AddItemToArray(accounts_json, account_json);
                }
            }
            for (jdx = 0; jdx < state2.no_accounts; jdx++) {
                if (writen_accounts[jdx] == 0) {
                    account2_ptr = &(state2.accounts[jdx]);
                    account_json = CuEVM::account::account_merge_json(
                        account1_ptr,
                        account2_ptr,
                        ACCOUNT_ALL_FLAG);
                    cJSON_AddItemToArray(accounts_json, account_json);
                }
            }
            delete[] writen_accounts;
            return state_json;
        }
    }
}
