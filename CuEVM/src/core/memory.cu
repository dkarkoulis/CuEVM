#include "hip/hip_runtime.h"
// CuEVM: CUDA Ethereum Virtual Machine implementation
// Copyright 2023 Stefan-Dan Ciocirlan (SBIP - Singapore Blockchain Innovation Programme)
// Author: Stefan-Dan Ciocirlan
// Data: 2023-11-30
// SPDX-License-Identifier: MIT

#include <CuEVM/utils/error_codes.cuh>
#include <CuEVM/core/memory.cuh>

namespace CuEVM {
  namespace memory {
    __host__ __device__ void evm_memory_t::print() const {
      printf("Memory data: \n");
      printf("Size: %d\n", size);
      printf("Memory cost: ");
      memory_cost.print();
      printf("\n");
      data.print();
    }

    __host__ cJSON* evm_memory_t::to_json() const {
      cJSON *json = cJSON_CreateObject();
      cJSON_AddItemToObject(json, "size", cJSON_CreateNumber(size));
      char *hex_string_ptr = memory_cost.to_hex();
      cJSON_AddStringToObject(json, "memory_cost", hex_string_ptr);
      delete[] hex_string_ptr;
      cJSON_AddItemToObject(json, "data", data.to_json());
      return json;
    }

    __host__ __device__ void evm_memory_t::get_memory_cost(
      ArithEnv &arith,
      bn_t &cost) const {
        cgbn_load(
          arith.env,
          cost,
          (CuEVM::cgbn_evm_word_t_ptr) &memory_cost
        );
    }

    __host__ __device__ void evm_memory_t::increase_memory_cost(
      ArithEnv &arith,
      const bn_t &memory_expansion_cost) {
        bn_t cuurent_cost;
        cgbn_load(
          arith.env,
          cuurent_cost,
          (CuEVM::cgbn_evm_word_t_ptr) &memory_cost
        );
        cgbn_add(
          arith.env,
          cuurent_cost,
          cuurent_cost,
          memory_expansion_cost
        );
        cgbn_store(
          arith.env,
          (CuEVM::cgbn_evm_word_t_ptr) &memory_cost,
          cuurent_cost
        );
      }

    __host__ __device__ int32_t evm_memory_t::allocate_pages(
      uint32_t new_size
    )  {
      if (new_size < data.size) {
        return ERROR_SUCCESS;
      }
      uint32_t new_page_count = (new_size / CuEVM::memory::page_size) + 1;
      return data.grow(new_page_count * CuEVM::memory::page_size, 1);
    }

    __host__ __device__ int32_t evm_memory_t::get_last_offset(
      ArithEnv &arith,
      const bn_t &index,
      const bn_t &length,
      uint32_t &offset
    ) const {
      int32_t overflow = 0;
      bn_t offset_bn;
      overflow = cgbn_add(arith.env, offset_bn, index, length);
      overflow |= arith.uint32_t_from_cgbn(offset, offset_bn);
      bn_t memory_size;
      overflow |= cgbn_add_ui32(arith.env, memory_size, offset_bn, 31);
      cgbn_div_ui32(arith.env, memory_size, memory_size, 32);
      overflow |= cgbn_mul_ui32(arith.env, offset_bn, memory_size, 32);
      overflow |= arith.uint32_t_from_cgbn(offset, offset_bn);

      return overflow;
    }

    __host__ __device__ int32_t evm_memory_t::grow(
      ArithEnv &arith,
      const bn_t &index,
      const bn_t &length) {
      uint32_t offset;
      if(get_last_offset(arith, index, length, offset) != 0) {
        return ERR_MEMORY_INVALID_OFFSET;
      }
      if (offset > size) {
        if(allocate_pages(offset) != 0) {
          return ERR_MEMORY_INVALID_ALLOCATION;
        }
        size = offset;
      }
      return ERROR_SUCCESS;
    }

    __host__ __device__ int32_t evm_memory_t::get(
      ArithEnv &arith,
      const bn_t &index,
      const bn_t &length,
      CuEVM::byte_array_t &data) {
      int32_t error_code = ERROR_SUCCESS;
      error_code = (cgbn_compare_ui32(arith.env, length, 0) < 0) ? ERR_MEMORY_INVALID_SIZE : error_code;
      error_code |= grow(arith, index, length);
      if (error_code == ERROR_SUCCESS) {
        uint32_t index_u32, length_u32;
        arith.uint32_t_from_cgbn(index_u32, index);
        arith.uint32_t_from_cgbn(length_u32, length);
        data = CuEVM::byte_array_t(this->data.data + index_u32, length_u32);
      } else {
        data = CuEVM::byte_array_t();
      }
      return error_code;
    }

  __host__ __device__ int32_t evm_memory_t::set(
      ArithEnv &arith,
      const CuEVM::byte_array_t &data,
      const bn_t &index,
      const bn_t &length) {

      int32_t error_code = ERROR_SUCCESS;
      error_code = (cgbn_compare_ui32(arith.env, length, 0) < 0) ? ERR_MEMORY_INVALID_SIZE : error_code;
      error_code |= grow(arith, index, length);
      if (error_code == ERROR_SUCCESS) {
        uint32_t index_u32, length_u32;
        arith.uint32_t_from_cgbn(index_u32, index);
        arith.uint32_t_from_cgbn(length_u32, length);

        if (length_u32 && data.size) {
          // memcpy(this->data.data + index_u32, data.data, min(length_u32,data.size));
          std::copy(data.data, data.data + min(length_u32,data.size), this->data.data + index_u32);
        }
      }
      return error_code;
    }

    __global__ void transfer_kernel(
      evm_memory_t *dst_instances,
      evm_memory_t *src_instances,
      uint32_t instance_count
    ) {
      uint32_t instance = blockIdx.x * blockDim.x + threadIdx.x;
      if (instance >= instance_count)
        return;

      if (src_instances[instance].data.size > 0)
      {
        memcpy(
          dst_instances[instance].data.data,
          src_instances[instance].data.data,
          src_instances[instance].data.size * sizeof(uint8_t)
        );
        delete[] src_instances[instance].data.data;
        src_instances[instance].data.data = nullptr;
      }
    }

    __host__ evm_memory_t *get_cpu(
        uint32_t count
    ) {
        return new evm_memory_t[count];
    }


    __host__ void cpu_free(
        evm_memory_t* instances,
        uint32_t count
    ) {
      delete[] instances;
    }

    __host__ evm_memory_t *get_gpu_from_cpu(
        evm_memory_t *cpu_instances,
        uint32_t count
    ) {
        evm_memory_t *gpu_instances, *tmp_cpu_instances;
        CUDA_CHECK(hipMalloc((void **)&gpu_instances, count * sizeof(evm_memory_t)));
        tmp_cpu_instances = new evm_memory_t[count];
        std::copy(cpu_instances, cpu_instances + count, tmp_cpu_instances);
        for (uint32_t i = 0; i < count; i++) {
            if (cpu_instances[i].data.size > 0) {
                CUDA_CHECK(hipMalloc((void **)&tmp_cpu_instances[i].data.data, cpu_instances[i].data.size));
                CUDA_CHECK(hipMemcpy(tmp_cpu_instances[i].data.data, cpu_instances[i].data.data, cpu_instances[i].data.size, hipMemcpyHostToDevice));
            }
        }
        CUDA_CHECK(hipMemcpy(tmp_cpu_instances, tmp_cpu_instances, count * sizeof(evm_memory_t), hipMemcpyHostToDevice));
        delete[] tmp_cpu_instances;
        return gpu_instances;
    }

    __host__ void gpu_free(
        evm_memory_t *gpu_instances,
        uint32_t count
    ) {
        evm_memory_t *tmp_cpu_instances = new evm_memory_t[count];
        CUDA_CHECK(hipMemcpy(tmp_cpu_instances, gpu_instances, count * sizeof(evm_memory_t), hipMemcpyDeviceToHost));
        for (uint32_t i = 0; i < count; i++) {
            if (tmp_cpu_instances[i].data.size > 0) {
                CUDA_CHECK(hipFree(tmp_cpu_instances[i].data.data));
            }
        }
        delete[] tmp_cpu_instances;
        CUDA_CHECK(hipFree(gpu_instances));
    }

    __host__ evm_memory_t *get_cpu_from_gpu(
      evm_memory_t *gpu_instances,
      uint32_t count
    ) {
      evm_memory_t *cpu_instances = new evm_memory_t[count];
      evm_memory_t *tmp_cpu_instances = new evm_memory_t[count];
      evm_memory_t* tmp_gpu_instances = nullptr;
      CUDA_CHECK(hipMemcpy(cpu_instances, gpu_instances, count * sizeof(evm_memory_t), hipMemcpyDeviceToHost));
      std::copy(cpu_instances, cpu_instances + count, tmp_cpu_instances);
      for (uint32_t idx = 0; idx < count; idx++) {
        if (tmp_cpu_instances[idx].size > 0)
        {
          CUDA_CHECK(hipMalloc(
            (void **)&tmp_cpu_instances[idx].data.data,
            sizeof(uint8_t) * tmp_cpu_instances[idx].data.size
          ));
        }
        else
        {
          tmp_cpu_instances[idx].data.data = nullptr;
        }
        tmp_cpu_instances[idx].size = tmp_cpu_instances[idx].size;
      }
      CUDA_CHECK(hipMalloc((void **)&tmp_gpu_instances, count * sizeof(evm_memory_t)));
      CUDA_CHECK(hipMemcpy(tmp_gpu_instances, tmp_cpu_instances, count * sizeof(evm_memory_t), hipMemcpyHostToDevice));
      delete[] tmp_cpu_instances;
      // 2. call the kernel to copy the memory between the gpu memories
      CuEVM::memory::transfer_kernel<<<count, 1>>>(tmp_gpu_instances, gpu_instances, count);
      CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipFree(gpu_instances));
      gpu_instances = tmp_gpu_instances;

      // 3. copy the gpu memories back in the cpu memories
      CUDA_CHECK(hipMemcpy(
        cpu_instances,
        gpu_instances,
        sizeof(evm_memory_t) * count,
        hipMemcpyDeviceToHost
      ));
      tmp_cpu_instances = new evm_memory_t[count];
      std::copy(cpu_instances, cpu_instances + count, tmp_cpu_instances);
      for (size_t idx = 0; idx < count; idx++)
      {
        if (tmp_cpu_instances[idx].data.size > 0)
        {
          tmp_cpu_instances[idx].data.data = new uint8_t[tmp_cpu_instances[idx].data.size];
          CUDA_CHECK(hipMemcpy(
            tmp_cpu_instances[idx].data.data,
            cpu_instances[idx].data.data,
            sizeof(uint8_t) * tmp_cpu_instances[idx].data.size,
            hipMemcpyDeviceToHost
          ));
        }
        else
        {
          tmp_cpu_instances[idx].data.data = nullptr;
        }
      }
      gpu_free(gpu_instances, count);
      std::copy(tmp_cpu_instances, tmp_cpu_instances + count, cpu_instances);
      delete[] tmp_cpu_instances;
      tmp_cpu_instances = NULL;
      return cpu_instances;
    }
  }
}